#include "hip/hip_runtime.h"
#include "Graph.h"
//#include <cutil_inline.h>
#include "/Volumes/Macintosh HD/Developer/NVIDIA/CUDA-7.5/samples/common/inc/hip/hip_runtime_api.h"
#include "/Volumes/Macintosh HD/Developer/NVIDIA/CUDA-7.5/samples/common/inc/helper_timer.h"
#include "/Volumes/Macintosh HD/Developer/NVIDIA/CUDA-7.5/samples/common/inc/helper_functions.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "utils.h"
#include "stats.h"
/**  TODO
* Convert Vertex to EulerEdge
*
*/
__global__ void componentStepInit(Vertex * v, unsigned int * D,  unsigned int* Q, unsigned int length){
	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	if( tid <length){
		//v[tid].vid;
		D[tid]=tid;
		Q[tid]=0;
	}
}

__global__ void componentStepOne_ShortCuttingP1(Vertex * v, unsigned  int * prevD, unsigned  int * curD, unsigned int * Q, unsigned int length, int s){
	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	if( tid <length){
		curD[tid] =prevD[prevD[tid]];
	}
}
__global__ void componentStepOne_ShortCuttingP2(Vertex * v, unsigned  int * prevD, unsigned  int * curD, unsigned int * Q, unsigned int length, int s){
	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	if( tid <length){
		if(curD[tid]!=prevD[tid]){
			Q[curD[tid]]=s;
		}
	}
}
//for edge
__global__ void componentStepTwo(Vertex * v,  unsigned int * prevD,  unsigned int * curD, unsigned int * Q, unsigned int length, unsigned  int s){
	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	int a;
	int valIdx;
	int val;

	if( tid <length ){
		//it will done for each edge 1
		if(curD[tid] == prevD[tid] && v[tid].n1<length ){
			if(curD[v[tid].n1] < curD[tid]){
				a=curD[tid]; valIdx= v[tid].n1; val=curD[valIdx];
				__syncthreads();
				atomicMin(curD+a,val);
				atomicExch(Q+val,s);
			}
		}

		//it will done for each edge 2
		if(curD[tid] == prevD[tid] && v[tid].n2<length){
			if(curD[v[tid].n2] < curD[tid]){
				a=curD[tid]; valIdx= v[tid].n2; val=curD[valIdx];
				__syncthreads();
				atomicMin(curD+a,val);
				atomicExch(Q+val,s);
				
				
			}
		}

	}
}

__global__ void componentStepTwoP1(Vertex * v,  unsigned int * prevD,  unsigned int * curD, unsigned int * Q,unsigned int * t1,unsigned int *val1 ,unsigned int * t2,unsigned int * val2, unsigned int length, unsigned  int s){
	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	
	if( tid <length ){
		t1[tid]=length;t2[tid]=length;
		//it will done for each edge 1
		if(curD[tid] == prevD[tid] && v[tid].n1<length ){
			if(curD[v[tid].n1] < curD[tid]){
				t1[tid]=curD[tid]; 
				val1[tid]=curD[v[tid].n1];
				
			}
		}

		//it will done for each edge 2
		if(curD[tid] == prevD[tid] && v[tid].n2<length){
			if(curD[v[tid].n2] < curD[tid]){
				t2[tid]=curD[tid]; 
				val2[tid]=curD[v[tid].n2];				
			}
		}

	}
}
__global__ void componentStepTwoP2(Vertex * v,  unsigned int * prevD,  unsigned int * curD, unsigned int * Q,unsigned int * t1,unsigned int *val1 ,unsigned int * t2,unsigned int * val2, unsigned int length, unsigned  int s){
	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	
	int a;	
	int val;

	if( tid <length ){	
		//it will done for each edge 1
		if(t1[tid]<length){			
			a=t1[tid];
			val=val1[tid];				
			atomicMin(curD+a,val);
			atomicExch(Q+val,s);
			
		}

		//it will done for each edge 2
		if(t2[tid]<length){			
			a=t2[tid];
			val=val2[tid];				
			atomicMin(curD+a,val);
			atomicExch(Q+val,s);		
		}
	}
}
//for edge
__global__ void componentStepThree(Vertex * v, unsigned int * prevD,unsigned  int * curD,unsigned int * Q,unsigned int length,unsigned int s){
	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	int a;
	int valIdx;
	int val;
	if( tid< length) {
		//it will be done for each edge 1
		if(curD[tid]==curD[curD[tid]] && Q[curD[tid]] < s && v[tid].n1<length){
			if( curD[tid] != curD[v[tid].n1] ){
				a=curD[tid]; valIdx=v[tid].n1;val= curD[valIdx];
				__syncthreads();
				atomicMin(curD+a,val);
				//curD[curD[tid]]= curD[v[tid].n1];
			}
		}
		//it will be done for each edge 2
		if(curD[tid]==curD[curD[tid]] && Q[curD[tid]] < s && v[tid].n2<length){
			if( curD[tid] != curD[v[tid].n2] ){
				a=curD[tid]; valIdx=v[tid].n2;val= curD[valIdx];
				__syncthreads();
				atomicMin(curD+a,val);
				//curD[curD[tid]]= curD[v[tid].n2];
			}
		}
		
	}
}
__global__ void componentStepThreeP1(Vertex * v, unsigned int * prevD,unsigned  int * curD,unsigned int * Q,unsigned int * t1,unsigned int *val1 ,unsigned int * t2,unsigned int * val2,unsigned int length,unsigned int s){
	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	if( tid< length) {
		t1[tid]=length; t2[tid]=length;
		//it will be done for each edge 1
		if(curD[tid]==curD[curD[tid]] && Q[curD[tid]] < s && v[tid].n1<length){
			if( curD[tid] != curD[v[tid].n1] ){				
				t1[tid]=curD[tid];
				val1[tid]= curD[v[tid].n1];				
			}
		}
		//it will be done for each edge 2
		if(curD[tid]==curD[curD[tid]] && Q[curD[tid]] < s && v[tid].n2<length){
			if( curD[tid] != curD[v[tid].n2] ){
				t2[tid]=curD[tid];
				val2[tid]= curD[v[tid].n2];
			}
		}
		
	}
}

__global__ void componentStepThreeP2(Vertex * v, unsigned int * prevD,unsigned  int * curD,unsigned int * Q,unsigned int * t1,unsigned int *val1 ,unsigned int * t2,unsigned int * val2,unsigned int length,unsigned int s){
	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	int a;	
	int val;
	if( tid< length) {
		//it will be done for each edge 1
		if(t1[tid]<length){			
			a=t1[tid]; 
			val= val1[tid];			
			atomicMin(curD+a,val);
				
		}
		//it will be done for each edge 2
		if(t2[tid]<length){			
			a=t2[tid];
			val= val2[tid];			
			atomicMin(curD+a,val);
				
		}		
	}
}

__global__ void componentStepFour(Vertex * v, unsigned  int * curD,unsigned int length){
	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	if( tid < length){
		unsigned val=curD[curD[tid]];
		__syncthreads();
		curD[tid]= val;
	}
	//curD[tid]= curD[curD[tid]];
}
__global__ void componentStepFourP1(Vertex * v, unsigned  int * curD,unsigned int * val1,unsigned int length){
	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	if( tid < length){
		val1[tid]=curD[curD[tid]];
	}	
}
__global__ void componentStepFourP2(Vertex * v, unsigned  int * curD,unsigned int * val1,unsigned int length){
	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	if( tid < length){
		curD[tid]= val1[tid];
	}
	
}
__global__ void componentStepFive(unsigned int * Q,unsigned int length,unsigned  int * sprimtemp,unsigned int s){
	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	if(tid <length) {
		if(Q[tid]==s){		
			atomicExch(sprimtemp,1);
			//*sprime=*sprimtemp+1;
		}
	}
}

extern "C"
void findComponentDevice(Vertex *d_v,unsigned int ** d_D, unsigned int length){
	
	
	unsigned int * d_prevD;
	unsigned int * d_Q;	 
	unsigned int * d_t1;
	unsigned int * d_t2;
	unsigned int * d_val1;
	unsigned int * d_val2;
	unsigned int sp;
	unsigned int * sptemp;
	unsigned int * d_sptemp;
	unsigned int s;
	unsigned int * temp;
	StopWatchInterface *timer = NULL;

	dim3 grid;
	dim3 block;

//	cutilCheckError(cutCreateTimer(&timer));
//	cutilCheckError(cutStartTimer(timer));

	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);
	
	getOptimalLaunchConfiguration(length,&grid,&block);
	
	allocateMemory((void**) &d_Q, length* sizeof(int));
	allocateMemory((void**) &d_t1, length* sizeof(int));
	allocateMemory((void**) &d_t2, length* sizeof(int));
	allocateMemory((void**) &d_val1, length* sizeof(int));
	allocateMemory((void**) &d_val2, length* sizeof(int));
	allocateMemory((void**) &d_prevD, length* sizeof(int));
	hipHostAlloc((void **)&sptemp, sizeof(int), hipHostMallocMapped);	
	hipHostGetDevicePointer((void **)&d_sptemp, (void *)sptemp, 0);	
	CheckCUDAError();
	//Initialize
	logMessage(LOG_LVL_DETAIL,"kernel: componentStepInit");
	componentStepInit<<<grid,block>>>(d_v,*d_D,d_Q,length);
	hipDeviceSynchronize();
	CheckCUDAError();

	s=1;
	sp=1;	
	while( s==sp)
	{		
		
		temp=*d_D;
		*d_D=d_prevD;
		d_prevD=temp;

		/**		componentStepOne_ShortCuttingP1		**/
		logMessage(LOG_LVL_DETAIL,"kernel: componentStepOne_ShortCuttingP1");
		componentStepOne_ShortCuttingP1<<<grid,block>>>(d_v, d_prevD, *d_D, d_Q,length,s);
		hipDeviceSynchronize();		
		CheckCUDAError();

		/**		componentStepOne_ShortCuttingP2		**/
		logMessage(LOG_LVL_DETAIL,"kernel: componentStepOne_ShortCuttingP2");
		componentStepOne_ShortCuttingP2<<<grid,block>>>(d_v, d_prevD, *d_D, d_Q,length,s);
		hipDeviceSynchronize();		
		CheckCUDAError();

		/**		componentStepTwoP1					**/
		logMessage(LOG_LVL_DETAIL,"kernel: componentStepTwoP1");
		componentStepTwoP1<<<grid,block>>>(d_v, d_prevD, *d_D, d_Q,d_t1,d_val1,d_t2,d_val2,length,s);
		hipDeviceSynchronize(); 
		CheckCUDAError();

		/**		componentStepTwoP2					**/
		logMessage(LOG_LVL_DETAIL,"kernel: componentStepTwoP2");
		componentStepTwoP2<<<grid,block>>>(d_v, d_prevD, *d_D, d_Q,d_t1,d_val1,d_t2,d_val2,length,s);
		hipDeviceSynchronize(); 
		CheckCUDAError();

		/**		componentStepThreeP1				**/
		logMessage(LOG_LVL_DETAIL,"kernel: componentStepThreeP1");
		componentStepThreeP1<<<grid,block>>>(d_v, d_prevD, *d_D, d_Q,d_t1,d_val1,d_t2,d_val2,length,s);
		hipDeviceSynchronize(); 
		CheckCUDAError();

		/**		componentStepThreeP2			**/
		logMessage(LOG_LVL_DETAIL,"kernel: componentStepThreeP2");
		componentStepThreeP2<<<grid,block>>>(d_v, d_prevD, *d_D, d_Q,d_t1,d_val1,d_t2,d_val2,length,s);
		hipDeviceSynchronize(); 
		CheckCUDAError();

		/**		componentStepFourP1				**/
		logMessage(LOG_LVL_DETAIL,"kernel: componentStepFourP1");
		componentStepFourP1<<<grid,block>>>(d_v,  *d_D,d_val1, length);
		hipDeviceSynchronize(); 
		CheckCUDAError();
		
		/**		componentStepFourP2				**/
		logMessage(LOG_LVL_DETAIL,"kernel: componentStepFourP2");
		componentStepFourP2<<<grid,block>>>(d_v,  *d_D,d_val1, length);
		hipDeviceSynchronize(); 
		CheckCUDAError();

		
		*sptemp=0;
		hipDeviceSynchronize(); 
		logMessage(LOG_LVL_DETAIL,"kernel: componentStepFive");
		componentStepFive<<<grid,block>>>(d_Q,length,d_sptemp,s);		
		hipDeviceSynchronize(); 
		CheckCUDAError();
		sp=sp+*sptemp;
		hipDeviceSynchronize(); 
		//printf("%d %d\n",s,*sptemp);
		s=s+1;
		//printData(*d_D,length);	
		
				
	}
//	printData(*d_D,length);
	deallocateMemory(d_t1);
	deallocateMemory(d_val1);
	deallocateMemory(d_t2);
	deallocateMemory(d_val2);
	deallocateMemory(d_Q);
	deallocateMemory(d_prevD);
	checkCudaErrors(hipHostFree(sptemp)); 	
	
//	cutilCheckError(cutStopTimer(timer));
//	setStatItem(TM_COMPONENT,cutGetTimerValue(timer));
//	cutilCheckError(cutDeleteTimer(timer));
	
	sdkStopTimer(&timer);
	float time = sdkGetTimerValue(&timer);
	setStatItem(TM_COMPONENT, time);
	sdkDeleteTimer(&timer);
}

extern "C"
void findComponent(Vertex *v,unsigned int * D, unsigned int length){
	
	unsigned int * d_curD;
	Vertex * d_v=NULL;

	checkCudaErrors( hipMalloc( (void**) &d_v, length* sizeof(Vertex)) );	
	checkCudaErrors( hipMalloc( (void**) &d_curD, length* sizeof(int) ));	
	
	checkCudaErrors( hipMemcpy(d_v, v, length*(sizeof(Vertex)), hipMemcpyHostToDevice) );
	findComponentDevice(d_v,&d_curD,length);

	checkCudaErrors( hipMemcpy(D, d_curD, length*(sizeof(int)), hipMemcpyDeviceToHost) );
	checkCudaErrors( hipFree(d_v) );	
	checkCudaErrors( hipFree(d_curD) );	
}

