#include "hip/hip_runtime.h"
__device__ __constant__ KEY_T lmerMask[] ={
    0x0000000000000003, 0x000000000000000F, 0x000000000000003F, 0x00000000000000FF, // 0   1   2   3
    0x00000000000003FF, 0x0000000000000FFF, 0x0000000000003FFF, 0x000000000000FFFF, // 4   5   6   7
    0x000000000003FFFF, 0x00000000000FFFFF, 0x00000000003FFFFF, 0x0000000000FFFFFF, // 8   9   10  11
    0x0000000003FFFFFF, 0x000000000FFFFFFF, 0x000000003FFFFFFF, 0x00000000FFFFFFFF, // 12  13  14  15
    0x00000003FFFFFFFF, 0x0000000FFFFFFFFF, 0x0000003FFFFFFFFF, 0x000000FFFFFFFFFF, // 16  17  18  19
    0x000003FFFFFFFFFF, 0x00000FFFFFFFFFFF, 0x00003FFFFFFFFFFF, 0x0000FFFFFFFFFFFF, // 20  21  22  23
    0x0003FFFFFFFFFFFF, 0x000FFFFFFFFFFFFF, 0x003FFFFFFFFFFFFF, 0x00FFFFFFFFFFFFFF, // 24  25  26  27
    0x03FFFFFFFFFFFFFF, 0x0FFFFFFFFFFFFFFF, 0x3FFFFFFFFFFFFFFF, 0xFFFFFFFFFFFFFFFF // 28  29  30  31
};

__device__ __constant__ unsigned char shifter[4] [4]=
{
		{0,0,0,0},
		{1,4,16,64},
		{2,8,32,128},
		{3,12,48,192},
};

__device__ __constant__ char  codeF[]={0,0,0,1,3,0,0,2};
__device__ __constant__ char  codeR[]={0,3,0,2,0,0,0,1};
__global__ void encodeLmerDevice(char  * buffer,
			//	const unsigned int buffSize,
			//	const unsigned int readLength,
				KEY_PTR lmers,
				const unsigned int lmerLength
				)
{
//    printf("in GPU");
	extern __shared__ char dnaRead[]; // MB: changed from 'read' to solve compile error
	const unsigned int tid=threadIdx.x;
	const unsigned int rOffset=(blockDim.x*blockDim.y*gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x) + (blockDim.x*threadIdx.y);
	KEY_T lmer=0;

	dnaRead[tid]=buffer[rOffset+tid];
	__syncthreads();

	for (unsigned int i = 0; i < 8; i++)    //calculate lmer
	{
	    lmer= (lmer<< 8) |	((KEY_T)(shifter[codeF[dnaRead[threadIdx.x+i*4]& 0x07]][3] |
							shifter[codeF[dnaRead[threadIdx.x+i*4+1]& 0x07]][2] |
							shifter[codeF[dnaRead[threadIdx.x+i*4+2]& 0x07]][1] |
							codeF[dnaRead[threadIdx.x+i*4+3] & 0x07]) ) ;
	}
	lmer = (lmer >> ((32 - lmerLength) << 1)) & lmerMask[lmerLength-1];
//    printf("%llu", lmer);
	lmers[rOffset+tid]=lmer;


}

__global__ void computeKmerDevice( 	KEY_PTR lmers,
				KEY_PTR pkmers,
				KEY_PTR skmers,
				KEY_T validBitMask
			){

	const unsigned int tid=(blockDim.x*blockDim.y*gridDim.x*blockIdx.y) +(blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	KEY_T lmer;
	//fetch lmer
	lmer=lmers[tid];
	//find prefix
	pkmers[tid]=LMER_PREFIX(lmer,validBitMask);
	//find suffix
	skmers[tid] = LMER_SUFFIX(lmer,validBitMask);
}

__global__ void encodeLmerComplementDevice(	char  * buffer,
				const unsigned int buffSize,
				const unsigned int readLength,
				KEY_PTR lmers,
				const unsigned int lmerLength
				)
{

	extern __shared__ char dnaRead[];//have to fix it
	const unsigned int tid=threadIdx.x;
	const unsigned int rOffset=(blockDim.x*blockDim.y*gridDim.x*blockIdx.y) +(blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y);
	KEY_T lmer=0;
	KEY_T temp=0;

	dnaRead[tid]=buffer[rOffset+tid];
	__syncthreads();
	dnaRead[tid]=codeR[dnaRead[tid] & 0x07];
	__syncthreads();
	for(unsigned int i =0; i< lmerLength; i++)
	{
		temp=((KEY_T)dnaRead[(tid+i)%blockDim.x]);
		lmer = (temp<<(i<<1)) | lmer;
	}
	lmers[rOffset+tid]=lmer;

}