#include "hip/hip_runtime.h"
#define CUDPP_STATIC_LIB
// #include "../nvidia_sdk/C/common/inc/cutil_inline.h"
#include "/Volumes/Macintosh HD/Developer/NVIDIA/CUDA-7.5/samples/common/inc/hip/hip_runtime_api.h" // lib above replaced w/this one at CUDA 5.0
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <stdio.h>      /* defines printf for tests */
#include <time.h>       /* defines time_t for timings in the test */
#include <math.h>

#ifdef linux
# include <endian.h>    /* attempt to define endianness */
#endif

//#include "debruijn.h"
#include "kmer.h"
#include "graph.h"
#include "cudpp.h"
#include "utils.h"
#include "common.h"
#include "gpuhash.h"
#include "gpuhash_device.h"

//#include "utils.cpp"

#if (defined(__BYTE_ORDER) && defined(__LITTLE_ENDIAN) && \
	__BYTE_ORDER == __LITTLE_ENDIAN) || \
	(defined(i386) || defined(__i386__) || defined(__i486__) || \
	defined(__i586__) || defined(__i686__) || defined(vax) || defined(MIPSEL))
# define HASH_LITTLE_ENDIAN 1
# define HASH_BIG_ENDIAN 0
#elif (defined(__BYTE_ORDER) && defined(__BIG_ENDIAN) && \
	__BYTE_ORDER == __BIG_ENDIAN) || \
	(defined(sparc) || defined(POWERPC) || defined(mc68000) || defined(sel))
# define HASH_LITTLE_ENDIAN 0
# define HASH_BIG_ENDIAN 1
#else
# define HASH_LITTLE_ENDIAN 0
# define HASH_BIG_ENDIAN 0
#endif

/*
 unsigned int _host_hash_h(KEY_T key, unsigned int bucketCount){
 return ((C0+C1*key)% LARGE_PRIME )% bucketCount;
 }


 VALUE_T getHashValue2Host(KEY_T key, TABLE_PTR T,unsigned int *bucketSize, unsigned int bucketCount){

 unsigned int bucket=_host_hash_h(key,bucketCount);
 unsigned int l=0;
 unsigned int r=bucketSize[bucket];
 unsigned int offset=bucket * BUCKET_SIZE;
 unsigned int mid=(l+r)>>1;
 while(l<r){
 mid =l+((r-l)/2);
 if( T[offset+(mid<<1)] <key) {
 l=mid+1;
 }else {
 r=mid;
 }
 }
 if(l < bucketSize[bucket] && T[offset+(l<<1)]==key){
 return T[offset+(l<<1)+1];
 }else {
 return MAX_INT;
 }
 }
 */

/***
 * Inline Printing Routine for l and e structures
 */
inline void printData(unsigned int * d_lstart, unsigned int * d_lcount,
		unsigned int * d_estart, unsigned int * d_ecount, unsigned int length) {

	unsigned int * h_lstart;
	unsigned int * h_lcount;
	unsigned int * h_estart;
	unsigned int * h_ecount;

	h_lstart = (unsigned int *) malloc(sizeof(unsigned int) * length);
	h_lcount = (unsigned int *) malloc(sizeof(unsigned int) * length);
	h_estart = (unsigned int *) malloc(sizeof(unsigned int) * length);
	h_ecount = (unsigned int *) malloc(sizeof(unsigned int) * length);

	checkCudaErrors(
			hipMemcpy(h_lstart, d_lstart, length * sizeof(unsigned int),
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_lcount, d_lcount, length * sizeof(unsigned int),
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_estart, d_estart, length * sizeof(unsigned int),
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_ecount, d_ecount, length * sizeof(unsigned int),
					hipMemcpyDeviceToHost));

	for (unsigned int i = 0; i < length; i++) {
		printf("[%4u]\t es:%4u\t ec:%4u\t ls:%4u\t lc:%4u\n", i, h_estart[i],
				h_ecount[i], h_lstart[i], h_lcount[i]);
	}

	free(h_lstart);
	free(h_lcount);
	free(h_estart);
	free(h_ecount);

}
//__global__ void debruijnCount(unsigned int k,unsigned long kmerCount, unsigned char * idata,unsigned int * icount,unsigned int * vcount, unsigned int * lcount,unsigned int * ecount,unsigned int  validBitMask){

/*
 *  This kernel works on each l-mer ,counting edges of the graph.
 */
__global__ void debruijnCount(KEY_PTR lmerKeys, /* lmer keys	*/
VALUE_PTR lmerValues, /* lmer frequency */
unsigned int lmerCount, /* total lmers */
KEY_PTR TK, /* Keys' pointer for Hash table*/
VALUE_PTR TV, /* Value pointer for Hash table*/
unsigned int * bucketSeed, /* bucketSize: size of each bucket (it should be renamed to bucketSize)*/
unsigned int bucketCount, /* total buckets */
unsigned int * lcount, /* leaving edge count array : OUT */
unsigned int * ecount, /* entering edge count array: OUT */
KEY_T validBitMask /* bit mask for K length encoded bits*/
) {

	unsigned int tid = (blockDim.x * blockDim.y * gridDim.x * blockIdx.y)
			+ (blockDim.x * blockDim.y * blockIdx.x)
			+ (blockDim.x * threadIdx.y) + threadIdx.x;
	if (tid < lmerCount) {
		KEY_T lmer = lmerKeys[tid];
		VALUE_T lmerValue = lmerValues[tid];
		KEY_T prefix = (lmer & (validBitMask << 2)) >> 2;
		KEY_T suffix = (lmer & validBitMask);

		KEY_T lomask = 3;
		VALUE_T prefixIndex = getHashValue(prefix, TK, TV, bucketSeed,
				bucketCount);
		VALUE_T suffixIndex = getHashValue(suffix, TK, TV, bucketSeed,
				bucketCount);
		KEY_T transitionTo = (lmer & lomask);
		KEY_T transitionFrom = ((lmer >> __popcll(validBitMask)) & lomask);
		//atomicAdd(lcount+(prefixIndex<<2 )+transition,lmerValue);
		//atomicAdd(ecount+(suffixIndex<<2)+transition,lmerValue);
		lcount[(prefixIndex << 2) + transitionTo] = lmerValue;
		ecount[(suffixIndex << 2) + transitionFrom] = lmerValue;
	}
}

/**
 * This is cpu version for same kernel. for Debugging purpose only
 */
void debruijnCountHost(KEY_PTR lmerKeys, VALUE_PTR lmerValues,
		unsigned int lmerCount, KEY_PTR TK, VALUE_PTR TV,
		unsigned int * bucketSeed, unsigned int bucketCount,
		unsigned int * lcount, unsigned int * ecount, KEY_T validBitMask,
		unsigned int bitCount, unsigned int tid) {

//	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	if (tid < lmerCount) {
		KEY_T lmer = lmerKeys[tid];
		VALUE_T lmerValue = lmerValues[tid];
		KEY_T prefix = (lmer & (validBitMask << 2)) >> 2;
		KEY_T suffix = (lmer & validBitMask);

		KEY_T lomask = 3;
		unsigned int b;
		VALUE_T prefixIndex = host_getHashValue(prefix, TK, TV, bucketSeed,
				bucketCount, &b);
		VALUE_T suffixIndex = host_getHashValue(suffix, TK, TV, bucketSeed,
				bucketCount, &b);
		KEY_T transitionTo = (lmer & lomask);
		KEY_T transitionFrom = ((lmer >> bitCount) & lomask);
		//atomicAdd(lcount+(prefixIndex<<2 )+transition,lmerValue);
		//atomicAdd(ecount+(suffixIndex<<2)+transition,lmerValue);
		if (lcount[(prefixIndex << 2) + transitionTo] > 0) {
			lcount[(prefixIndex << 2) + transitionTo] = lmerValue;
		} else {
			lcount[(prefixIndex << 2) + transitionTo] = lmerValue;
		}
		if (ecount[(suffixIndex << 2) + transitionFrom] > 0) {
			ecount[(suffixIndex << 2) + transitionFrom] = lmerValue;
		} else {
			ecount[(suffixIndex << 2) + transitionFrom] = lmerValue;
		}

	}
}

/*
 * stub for debruijnCountHost for debugging purpose
 */
void verifyDebruijnCountHost(KEY_PTR d_lmerKeys, VALUE_PTR d_lmerValues,
		unsigned int lmerCount, KEY_PTR d_TK, VALUE_PTR d_TV,
		unsigned int * d_bucketSeed, unsigned int bucketCount,
		unsigned int * d_lcount, unsigned int * d_ecount, KEY_T validBitMask,
		unsigned int kmerCount) {

	KEY_PTR h_lmerKeys;
	VALUE_PTR h_lmerValues;
	KEY_PTR h_TK;
	VALUE_PTR h_TV;
	unsigned int * h_bucketSeed;
	unsigned int * h_lcount;
	unsigned int * h_ecount;
	unsigned int * hq_lcount;
	unsigned int * hq_ecount;

	h_lmerKeys = (KEY_PTR) malloc(lmerCount * sizeof(KEY_T));
	h_lmerValues = (VALUE_PTR) malloc(lmerCount * sizeof(VALUE_T));
	h_TK = (KEY_PTR) malloc(bucketCount * BUCKET_KEY_SIZE);
	h_TV = (VALUE_PTR) malloc(bucketCount * BUCKET_VALUE_SIZE);
	h_bucketSeed = (unsigned int *) malloc(bucketCount * sizeof(unsigned int));
	h_lcount = (unsigned int *) malloc(4 * kmerCount * sizeof(unsigned int));
	h_ecount = (unsigned int *) malloc(4 * kmerCount * sizeof(unsigned int));
	hq_lcount = (unsigned int *) malloc(4 * kmerCount * sizeof(unsigned int));
	hq_ecount = (unsigned int *) malloc(4 * kmerCount * sizeof(unsigned int));

	checkCudaErrors(
			hipMemcpy(h_lmerKeys, d_lmerKeys, lmerCount * KEY_SIZE,
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_lmerValues, d_lmerValues, lmerCount * VALUE_SIZE,
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_TK, d_TK, bucketCount * BUCKET_KEY_SIZE,
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_TV, d_TV, bucketCount * BUCKET_VALUE_SIZE,
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_bucketSeed, d_bucketSeed,
					bucketCount * sizeof(unsigned int),
					hipMemcpyDeviceToHost));

	checkCudaErrors(
			hipMemcpy(hq_lcount, d_lcount,
					4 * kmerCount * sizeof(unsigned int),
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(hq_ecount, d_ecount,
					4 * kmerCount * sizeof(unsigned int),
					hipMemcpyDeviceToHost));

	memset(h_lcount, 0, 4 * kmerCount * sizeof(unsigned int));
	memset(h_ecount, 0, 4 * kmerCount * sizeof(unsigned int));
	unsigned int bitCount = 0;
	KEY_T bit = 1;
	while (bit != 0) {
		if (bit & validBitMask)
			bitCount++;
		bit = bit << 1;
	}
	unsigned int edgesCount = 0;
	for (unsigned int i = 0; i < lmerCount; i++) {
		debruijnCountHost(h_lmerKeys, h_lmerValues, lmerCount, h_TK, h_TV,
				h_bucketSeed, bucketCount, h_lcount, h_ecount, validBitMask,
				bitCount, i);
		edgesCount += h_lmerValues[i];
	}
	unsigned int esum = 0;
	unsigned int qesum = 0;
	unsigned int lsum = 0;
	unsigned int qlsum = 0;
	unsigned int ei = 0;
	unsigned int li = 0;
	unsigned int qei = 0;
	unsigned int qli = 0;
	for (int j = 0; j < 4 * kmerCount; j++) {
		esum += h_ecount[j];
		lsum += h_lcount[j];
		qesum += hq_ecount[j];
		qlsum += hq_lcount[j];
		if (esum > edgesCount && ei < 1)
			ei = j;
		if (lsum > edgesCount && li < 1)
			li = j;
		if (qesum > edgesCount && qei < 1)
			qei = j;
		if (qlsum > edgesCount && qli < 1)
			qli = j;
	}
	printf(
			"lmerCount: %u, esum: %u, lsum: %u, ei: %u li:%u \n qesum:%u , qlsum:%u \n",
			edgesCount, esum, lsum, ei, li, qesum, qlsum);

	unsigned int enc = 0;
	unsigned int lnc = 0;
	for (unsigned int k = 0; k < 4 * kmerCount; k++) {
		if (h_lcount[k] != hq_lcount[k])
			lnc++;
		if (h_ecount[k] != hq_ecount[k])
			enc++;
	}
	printf("enc: %u,  lnc:%u \n", enc, lnc);
	free(h_lmerValues);
	free(h_TK);
	free(h_TV);
	free(h_bucketSeed);
	free(h_lcount);
	free(h_ecount);
	free(hq_lcount);
	free(hq_ecount);

}

/**
 * CPU prefix scan
 */
void prefixScan(unsigned int * h_out, unsigned int * h_in, unsigned int length,
		bool inclusive) {

	memset(h_out, 0, length * sizeof(unsigned int));
	/*calculate gold*/
	if (inclusive) {
		h_out[0] = h_in[0];
	} else {
		h_out[0] = 0;
	}
	for (unsigned int i = 1; i < length; i++) {
		h_out[i] = h_out[i - 1] + h_in[i - (inclusive ? 0 : 1)];
	}

}
/*
 * prefix sum validator
 **/
void validatePrefixScan(unsigned int * d_output, unsigned int * d_input,
		unsigned int length, bool inclusive) {

	unsigned int * h_input;
	unsigned int * h_output;
	unsigned int * hq_output;

	h_input = (unsigned int *) malloc(length * sizeof(unsigned int));
	h_output = (unsigned int *) malloc(length * sizeof(unsigned int));
	hq_output = (unsigned int *) malloc(length * sizeof(unsigned int));

	checkCudaErrors(
			hipMemcpy(h_input, d_input, length * sizeof(unsigned int),
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(hq_output, d_output, length * sizeof(unsigned int),
					hipMemcpyDeviceToHost));

	/*
	 memset(h_output,0,length*sizeof(unsigned int));
	 if(inclusive)
	 {	h_output[0]=h_input[0];}
	 else
	 {	h_output[0]=0;	}
	 for(unsigned int i=1;i<length;i++){
	 h_output[i]=h_output[i-1]+h_input[i-(inclusive?0:1)];
	 }
	 */
	prefixScan(h_output, h_input, length, inclusive);
	/*compare*/
	for (unsigned int j = 0; j < length; j++) {
		if (h_output[j] != hq_output[j]) {
			printf("differnce at index:%u is gold:%u, cudpp\n", j, h_output[j], hq_output[j]);
		}
	}

	free(h_input);
	free(h_output);
	free(hq_output);
}
/*
 *  This kernel works on a k-mer (l-1mer) which are vertices of the graph.
 */
__global__ void setupVertices(KEY_PTR kmerKeys, unsigned int kmerCount,
		KEY_PTR TK, VALUE_PTR TV, unsigned int * bucketSeed,
		unsigned int bucketCount, EulerVertex * ev, unsigned int * lcount,
		unsigned int * loffset, unsigned int * ecount, unsigned int * eoffset) {
	unsigned int tid = (blockDim.x * blockDim.y * gridDim.x * blockIdx.y)
			+ (blockDim.x * blockDim.y * blockIdx.x)
			+ (blockDim.x * threadIdx.y) + threadIdx.x;
	if (tid < kmerCount) {
		KEY_T key = kmerKeys[tid];
		VALUE_T index = getHashValue(key, TK, TV, bucketSeed, bucketCount);
		;
		ev[index].vid = key;
		ev[index].lp = loffset[(index << 2)];
		ev[index].lcount = lcount[(index << 2)] + lcount[(index << 2) + 1]
				+ lcount[(index << 2) + 2] + lcount[(index << 2) + 3];
		ev[index].ep = eoffset[(index << 2)];
		ev[index].ecount = ecount[(index << 2)] + ecount[(index << 2) + 1]
				+ ecount[(index << 2) + 2] + ecount[(index << 2) + 3];
	}
}
void setupVerticesHost(KEY_PTR kmerKeys, unsigned int kmerCount, KEY_PTR TK,
		VALUE_PTR TV, unsigned int * bucketSeed, unsigned int bucketCount,
		EulerVertex * ev, unsigned int * lcount, unsigned int * loffset,
		unsigned int * ecount, unsigned int * eoffset, unsigned int tid) {
//	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	if (tid < kmerCount) {
		KEY_T key = kmerKeys[tid];
		unsigned int bucket;
		VALUE_T index = host_getHashValue(key, TK, TV, bucketSeed, bucketCount,
				&bucket);
		;
		ev[index].vid = key;
		ev[index].lp = loffset[(index << 2)];
		ev[index].lcount = lcount[(index << 2)] + lcount[(index << 2) + 1]
				+ lcount[(index << 2) + 2] + lcount[(index << 2) + 3];
		ev[index].ep = eoffset[(index << 2)];
		ev[index].ecount = ecount[(index << 2)] + ecount[(index << 2) + 1]
				+ ecount[(index << 2) + 2] + ecount[(index << 2) + 3];
	}
}

/* 
 *  This kernel works on an l-mer, which represents an edge
 *  in the debruijn Graph.
 */
__global__ void setupEdges( KEY_PTR  lmerKeys,  VALUE_PTR  lmerValues,
		 unsigned int *  lmerOffsets, const unsigned int lmerCount,
		 KEY_PTR  TK, VALUE_PTR  TV, unsigned int *  bucketSeed,
		const unsigned int bucketCount, unsigned int *  l,
		 unsigned int *  e, EulerEdge *  ee,
		 unsigned int *  loffsets, unsigned int *  eoffsets,
		const KEY_T validBitMask) {

	unsigned int tid = (blockDim.x * blockDim.y * gridDim.x * blockIdx.y)
			+ (blockDim.x * blockDim.y * blockIdx.x)
			+ (blockDim.x * threadIdx.y) + threadIdx.x;
	if (tid < lmerCount) {
		KEY_T lmer = lmerKeys[tid];
		VALUE_T lmerValue = lmerValues[tid];
		KEY_T prefix = (lmer & (validBitMask << 2)) >> 2;
		KEY_T suffix = (lmer & validBitMask);
		KEY_T lomask = 3;
		//prefix and suffix index must be less than kmer count
		VALUE_T prefixIndex = getHashValue(prefix, TK, TV, bucketSeed,
				bucketCount);
		VALUE_T suffixIndex = getHashValue(suffix, TK, TV, bucketSeed,
				bucketCount);
		KEY_T transitionTo = (lmer & lomask);
		KEY_T transitionFrom = ((lmer >> __popcll(validBitMask)) & lomask);
		unsigned int loffset = loffsets[(prefixIndex << 2) + transitionTo];
		unsigned int eoffset = eoffsets[(suffixIndex << 2) + transitionFrom];

		unsigned int lmerOffset = lmerOffsets[tid];
		for (unsigned int i = 0; i < lmerValue; i++) {

			ee[lmerOffset].eid =lmerOffset;
			ee[lmerOffset].v1 = prefixIndex;
			ee[lmerOffset].v2 = suffixIndex;
			// lmerOffset;
			ee[lmerOffset].s = lmerValues[lmerCount - 1]
					+ lmerOffsets[lmerCount - 1];

			l[loffset] = lmerOffset;
			e[eoffset] = lmerOffset;
			loffset++;
			eoffset++;
			lmerOffset++;
		}
	}
}
void setupEdgesHost(KEY_PTR const lmerKeys, VALUE_PTR const lmerValues,
		unsigned int * const lmerOffsets, const unsigned int lmerCount,
		KEY_PTR const TK, VALUE_PTR const TV, unsigned int * const bucketSeed,
		const unsigned int bucketCount, unsigned int * const l,
		unsigned int * const e, EulerEdge * const ee,
		unsigned int * const loffsets, unsigned int * const eoffsets,
		const KEY_T validBitMask, const unsigned int tid) {

//	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	if (tid < lmerCount) {
		KEY_T lmer = lmerKeys[tid];
		VALUE_T lmerValue = lmerValues[tid];
		KEY_T prefix = (lmer & (validBitMask << 2)) >> 2;
		KEY_T suffix = (lmer & validBitMask);
		KEY_T lomask = 3;
		unsigned int bucket;
		//prefix and suffix index must be less than kmer count
		VALUE_T prefixIndex = host_getHashValue(prefix, TK, TV, bucketSeed,
				bucketCount, &bucket);
		VALUE_T suffixIndex = host_getHashValue(suffix, TK, TV, bucketSeed,
				bucketCount, &bucket);
		KEY_T transitionTo = (lmer & lomask);
		KEY_T transitionFrom = ((lmer >> 16) & lomask);
		unsigned int loffset = loffsets[(prefixIndex << 2) + transitionTo];
		unsigned int eoffset = eoffsets[(suffixIndex << 2) + transitionFrom];

		unsigned int lmerOffset = lmerOffsets[tid];
		for (int i = 0; i < lmerValue; i++) {
			ee[lmerOffset].eid = lmerOffset;
			ee[lmerOffset].v1 = prefixIndex;
			ee[lmerOffset].v2 = suffixIndex;
			ee[lmerOffset].s = lmerValues[lmerCount - 1]
					+ lmerOffsets[lmerCount - 1];

			l[loffset] = lmerOffset;
			e[eoffset] = lmerOffset;
			loffset++;
			eoffset++;
			lmerOffset++;
		}
	}
}

void verifySetupEdges(KEY_PTR d_lmerKeys, VALUE_PTR d_lmerValues,
		unsigned int * d_lmerOffsets, const unsigned int lmerCount,
		KEY_PTR d_TK, VALUE_PTR d_TV, unsigned int * d_bucketSeed,
		const unsigned int bucketCount, unsigned int * d_l, unsigned int * d_e,
		EulerEdge * d_ee, unsigned int * d_lcount, unsigned int * d_loffsets,
		unsigned int * d_ecount, unsigned int * d_eoffsets,
		unsigned int kmerCount, unsigned int ecount, const KEY_T validBitMask) {

	KEY_PTR h_lmerKeys;
	VALUE_PTR h_lmerValues;
	unsigned int * h_lmerOffsets;
	KEY_PTR h_TK;
	VALUE_PTR h_TV;
	unsigned int * h_bucketSeed;
	unsigned int * h_l;
	unsigned int * h_e;
	EulerEdge * h_ee;
	unsigned int * h_loffsets;
	unsigned int * h_lcount;
	unsigned int * h_eoffsets;
	unsigned int * h_ecount;

	h_lmerKeys = (KEY_PTR) malloc(lmerCount * KEY_SIZE);
	h_lmerValues = (VALUE_PTR) malloc(lmerCount * VALUE_SIZE);
	h_lmerOffsets = (unsigned int *) malloc(lmerCount * sizeof(unsigned int));
	h_TK = (KEY_PTR) malloc(bucketCount * BUCKET_KEY_SIZE);
	h_TV = (VALUE_PTR) malloc(bucketCount * BUCKET_VALUE_SIZE);
	h_bucketSeed = (unsigned int *) malloc(bucketCount * sizeof(unsigned int));
	h_bucketSeed = (unsigned int *) malloc(bucketCount * sizeof(unsigned int));
	h_l = (unsigned int *) malloc(ecount * sizeof(unsigned int));
	h_e = (unsigned int *) malloc(ecount * sizeof(unsigned int));
	h_ee = (EulerEdge *) malloc(ecount * sizeof(EulerEdge));
	h_loffsets = (unsigned int *) malloc(kmerCount * 4 * sizeof(unsigned int));
	h_eoffsets = (unsigned int *) malloc(kmerCount * 4 * sizeof(unsigned int));
	h_lcount = (unsigned int *) malloc(kmerCount * 4 * sizeof(unsigned int));
	h_ecount = (unsigned int *) malloc(kmerCount * 4 * sizeof(unsigned int));

	checkCudaErrors(
			hipMemcpy(h_lmerKeys, d_lmerKeys, lmerCount * KEY_SIZE,
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_lmerValues, d_lmerValues, lmerCount * VALUE_SIZE,
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_lmerOffsets, d_lmerOffsets,
					lmerCount * sizeof(unsigned int), hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_TK, d_TK, bucketCount * BUCKET_KEY_SIZE,
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_TV, d_TV, bucketCount * BUCKET_VALUE_SIZE,
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_bucketSeed, d_bucketSeed,
					bucketCount * sizeof(unsigned int),
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_ee, d_ee, ecount * sizeof(EulerEdge),
					hipMemcpyDeviceToHost));

	checkCudaErrors(
			hipMemcpy(h_loffsets, d_loffsets,
					kmerCount * 4 * sizeof(unsigned int),
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_eoffsets, d_eoffsets,
					kmerCount * 4 * sizeof(unsigned int),
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_lcount, d_lcount, kmerCount * 4 * sizeof(unsigned int),
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_ecount, d_ecount, kmerCount * 4 * sizeof(unsigned int),
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_l, d_l, ecount * sizeof(unsigned int),
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_e, d_e, ecount * sizeof(unsigned int),
					hipMemcpyDeviceToHost));

	printf(".....diff....\n");
	for (unsigned int j = 0; j < (4 * kmerCount) - 1; j++) {
		if (h_lcount[j] != h_loffsets[j + 1] - h_loffsets[j]) {
			printf(" lcount mismatch j:[%u] lcount:[%u] diff:[%u]\n", j,
					h_lcount[j], h_loffsets[j + 1] - h_loffsets[j]);
		}
		if (h_ecount[j] != h_eoffsets[j + 1] - h_eoffsets[j]) {
			printf(" ecount mismatch j:[%u] ecount:[%u] diff:[%u]\n", j,
					h_ecount[j], h_eoffsets[j + 1] - h_eoffsets[j]);
		}
	}

	/*	for(unsigned int k=0;k<4*kmerCount;k++){
	 printf("[%u]:  loffset[%u] ,lcount[%u] ,eoffset[%u], ecount[%u]\n",k,h_loffsets[k],h_lcount[k],h_eoffsets[k],h_ecount[k]);
	 }
	 */
	for (int i = 0; i < lmerCount; i++) {
		setupEdgesHost(h_lmerKeys, h_lmerValues, h_lmerOffsets, lmerCount, h_TK,
				h_TV, h_bucketSeed, bucketCount, h_l, h_e, h_ee, h_loffsets,
				h_eoffsets, validBitMask, i);
	}

	free(h_lmerKeys);
	free(h_lmerValues);
	free(h_lmerOffsets);
	free(h_TK);
	free(h_TV);
	free(h_bucketSeed);
	free(h_l);
	free(h_e);
	free(h_ee);
	free(h_loffsets);
	free(h_eoffsets);
	free(h_lcount);
	free(h_ecount);

}

void verifyleOffsets(unsigned int * d_lOffsets, unsigned int * d_lcount,
		unsigned int * d_eOffsets, unsigned int * d_ecount, unsigned int length,
		unsigned int ecount) {

	unsigned int * h_lOffsets;
	unsigned int * h_eOffsets;
	unsigned int * h_lcount;
	unsigned int * h_ecount;

	h_lOffsets = (unsigned int*) malloc(length * sizeof(unsigned int));
	h_eOffsets = (unsigned int *) malloc(length * sizeof(unsigned int));
	h_lcount = (unsigned int*) malloc(length * sizeof(unsigned int));
	h_ecount = (unsigned int *) malloc(length * sizeof(unsigned int));

	checkCudaErrors(
			hipMemcpy(h_lOffsets, d_lOffsets, length * sizeof(unsigned int),
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_eOffsets, d_eOffsets, length * sizeof(unsigned int),
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_lcount, d_lcount, length * sizeof(unsigned int),
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_ecount, d_ecount, length * sizeof(unsigned int),
					hipMemcpyDeviceToHost));
	unsigned int esum = 0;
	unsigned int lsum = 0;
	for (unsigned int t = 0; t < length; t++) {
		esum += h_ecount[t];
		lsum += h_lcount[t];
	}
	printf("esum : %u , lsum : %u \n", esum, lsum);
	unsigned int incorrectTotal = 0;
	for (unsigned int i = 0; i < length; i++) {
		if (h_lOffsets[i] > ecount || h_lOffsets[i] + h_lcount[i] > ecount) {
			incorrectTotal++;
			printf("incorrect l @ %u,  value %u\n",i,h_lOffsets[i]);
		}
		if (h_eOffsets[i] > ecount || h_eOffsets[i] + h_ecount[i] > ecount) {
			incorrectTotal++;
			printf("incorrect e @ %u,  value %u\n",i,h_eOffsets[i]);
		}
	}
	free(h_lOffsets);
	free(h_eOffsets);
	free(h_lcount);
	free(h_ecount);
}
extern "C" void constructDebruijnGraphGold(unsigned int * ecount,
		KEY_PTR h_lmerKeys, //in lmer keys
		VALUE_PTR h_lmerValues, //in lmer values
		unsigned int lmerCount, //in total lmers
		KEY_PTR h_kmerKeys, //in
		unsigned long kmerCount, //in  total kmers
		unsigned int l, //in k
		KEY_PTR h_TK, VALUE_PTR h_TV, unsigned int * h_bucketSeed,
		unsigned int bucketCount, EulerVertex ** h_ev, //out
		unsigned int ** h_l, //out
		unsigned int ** h_e, //out
		EulerEdge ** h_ee //out
		) { //out

	dim3 grid;
	dim3 block;

	unsigned int * h_lcount;
	unsigned int * h_lstart;
	unsigned int * h_ecount;
	unsigned int * h_estart;
	unsigned int * h_lmerOffsets;

	unsigned int memsize;
	KEY_T validBitMask = 0;
	//unsigned int timerGPU = 0;	
	unsigned int k = l - 1;

	//cutilCheckError(cutCreateTimer(&timerGPU));

	memsize = (kmerCount) * sizeof(unsigned int) * 4; // 4-tuple for each kmer

	h_lcount = (unsigned int *) malloc(memsize);
	h_lstart = (unsigned int *) malloc(memsize);
	h_estart = (unsigned int *) malloc(memsize);
	h_ecount = (unsigned int *) malloc(memsize);
	h_lmerOffsets = (unsigned int*) malloc(lmerCount * VALUE_SIZE);

	for (unsigned int i = 0; i < k * 2; i++) {
		validBitMask = (validBitMask << 1) | 1;
	}

	unsigned int bitCount = 0;
	KEY_T bit = 1;
	while (bit != 0) {
		if (bit & validBitMask)
			bitCount++;
		bit = bit << 1;
	}
	memset(h_lcount, 0, sizeof(unsigned int) * 4 * kmerCount);
	memset(h_ecount, 0, sizeof(unsigned int) * 4 * kmerCount);

	//verifyDebruijnCountHost(d_lmerKeys,d_lmerValues,lmerCount,d_TK,d_TV,d_bucketSeed,bucketCount,d_lcount,d_ecount,validBitMask,kmerCount);
	for (unsigned int tid = 0; tid < lmerCount; tid++) {
		debruijnCountHost(h_lmerKeys, h_lmerValues, lmerCount, h_TK, h_TV,
				h_bucketSeed, bucketCount, h_lcount, h_ecount, validBitMask,
				bitCount, tid);
	}

	/* we need to perform pre-fix scan on , lcount, ecount, lmerValues,
	 * lcount and ecount has equal number of elements ,4*kmercount
	 * lmer has lmerCount elements, choose whichever is larger
	 */

	memset(h_lstart, 0, sizeof(unsigned int) * 4 * kmerCount);
	memset(h_estart, 0, sizeof(unsigned int) * 4 * kmerCount);
	memset(h_lmerOffsets, 0, sizeof(unsigned int) * lmerCount);

	prefixScan(h_lstart, h_lcount, 4 * kmerCount, false);
	prefixScan(h_estart, h_ecount, 4 * kmerCount, false);

	prefixScan(h_lmerOffsets, h_lmerValues, lmerCount, false);

	/*
	 unsigned int buffer[2];
	 readData(buffer,d_lmerOffsets+lmerCount-1,1,sizeof(unsigned int));
	 readData(buffer+1,d_lmerValues+lmerCount-1,1,sizeof(unsigned int));
	 *ecount=buffer[0]+buffer[1];
	 */
	*ecount = h_lmerOffsets[lmerCount - 1] + h_lmerValues[lmerCount - 1];

	*h_ev = (EulerVertex *) malloc(sizeof(EulerVertex) * (kmerCount));
	*h_l = (unsigned int *) malloc(sizeof(unsigned int) * (*ecount));
	*h_e = (unsigned int *) malloc(sizeof(unsigned int) * (*ecount));
	*h_ee = (EulerEdge *) malloc(sizeof(EulerEdge) * (*ecount));
	memset(*h_e, 0, sizeof(unsigned int) * (*ecount));
	memset(*h_l, 0, sizeof(unsigned int) * (*ecount));

//	getOptimalLaunchConfiguration(kmerCount,&grid,&block);
	for (unsigned int tid = 0; tid < kmerCount; tid++) {
		setupVerticesHost(h_kmerKeys, kmerCount, h_TK, h_TV, h_bucketSeed,
				bucketCount, *h_ev, h_lcount, h_lstart, h_ecount, h_estart,
				tid);
	}

	//getOptimalLaunchConfiguration(lmerCount,&grid,&block);
	for (unsigned int tid = 0; tid < lmerCount; tid++) {
		setupEdgesHost(h_lmerKeys, h_lmerValues, h_lmerOffsets, lmerCount, h_TK,
				h_TV, h_bucketSeed, bucketCount, *h_l, *h_e, *h_ee, h_lstart,
				h_estart, validBitMask, tid);
	}

	free(h_lmerOffsets);
	free(h_lcount);
	free(h_lstart);
	free(h_estart);
	free(h_ecount);

}
//extern "C" 
void constructDebruijnGraphDevice(unsigned int * ecount,
		KEY_PTR d_lmerKeys, //in lmer keys
		VALUE_PTR d_lmerValues, //in lmer values
		unsigned int lmerCount, //in total lmers
		KEY_PTR d_kmerKeys, //in
		unsigned long kmerCount, //in  total kmers
		unsigned int l, //in k
		KEY_PTR d_TK, 
		VALUE_PTR d_TV, 
		unsigned int * d_bucketSeed,
		unsigned int bucketCount, 
		EulerVertex ** d_ev, //out
		unsigned int ** d_l, //out
		unsigned int ** d_e, //out
		EulerEdge ** d_ee //out
		) { //out

	dim3 grid;
	dim3 block;

	unsigned int * d_lcount;
	unsigned int * d_lstart;
	unsigned int * d_ecount;
	unsigned int * d_estart;
	unsigned int * d_lmerOffsets;

	unsigned int mem_size;
	KEY_T validBitMask = 0;
	//unsigned int timerGPU = 0;	
	unsigned int k = l - 1;

	//cutilCheckError(cutCreateTimer(&timerGPU));

	mem_size = (kmerCount) * sizeof(unsigned int) * 4; // 4-tuple for each kmer

	allocateMemory((void**) &d_lcount, mem_size);
	allocateMemory((void**) &d_lstart, mem_size);
	allocateMemory((void**) &d_estart, mem_size);
	allocateMemory((void**) &d_ecount, mem_size);
	allocateMemory((void**) &d_lmerOffsets, lmerCount * VALUE_SIZE);

	for (unsigned int i = 0; i < k * 2; i++) {
		validBitMask = (validBitMask << 1) | 1;
	}

	logMessage(LOG_LVL_DETAIL,"deb bit mask %lu\n",validBitMask);
	logMessage(LOG_LVL_DETAIL, "kernel: debruijnCount");
	getOptimalLaunchConfiguration(lmerCount, &grid, &block);
	//verifyDebruijnCountHost(d_lmerKeys,d_lmerValues,lmerCount,d_TK,d_TV,d_bucketSeed,bucketCount,d_lcount,d_ecount,validBitMask,kmerCount);
	debruijnCount<<<grid,block>>>(d_lmerKeys,d_lmerValues,lmerCount,d_TK,d_TV,d_bucketSeed,bucketCount,d_lcount,d_ecount,validBitMask);
	CheckCUDAError();
	//verifyDebruijnCountHost(d_lmerKeys,d_lmerValues,lmerCount,d_TK,d_TV,d_bucketSeed,bucketCount,d_lcount,d_ecount,validBitMask,kmerCount);

	/* we need to perform pre-fix scan on , lcount, ecount, lmerValues,
	 * lcount and ecount has equal number of elements ,4*kmercount
	 * lmer has lmerCount elements, choose whichever is larger
	 */

//	unsigned int maxElements=(lmerCount>4*kmerCount)?lmerCount:4*kmerCount;
	CUDPPConfiguration configKmer;
	configKmer.op = CUDPP_ADD;
	configKmer.datatype = CUDPP_UINT;
	configKmer.algorithm = CUDPP_SCAN;
	configKmer.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;

	CUDPPHandle scanplanKmer = 0;
	cudppPlan(&scanplanKmer, configKmer, 4 * kmerCount, 1, 0);
	CheckCUDAError();

	cudppScan(scanplanKmer, d_lstart, d_lcount, 4 * kmerCount);
	cudppScan(scanplanKmer, d_estart, d_ecount, 4 * kmerCount);
	cudppDestroyPlan(scanplanKmer);

	CUDPPConfiguration configLmer;
	configLmer.op = CUDPP_ADD;
	configLmer.datatype = CUDPP_UINT;
	configLmer.algorithm = CUDPP_SCAN;
	configLmer.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;

	CUDPPHandle scanplanLmer = 0;
	CUDPPResult result = cudppPlan(&scanplanLmer, configLmer, lmerCount, 1, 0);
	CheckCUDAError();

	cudppScan(scanplanLmer, d_lmerOffsets, d_lmerValues, lmerCount);
	cudppDestroyPlan(scanplanLmer);

	//validatePrefixScan(d_lstart,d_lcount,4*kmerCount,false);
	//validatePrefixScan(d_estart,d_ecount,4*kmerCount,false);
	//validatePrefixScan(d_lmerOffsets,d_lmerValues,lmerCount,false);

	unsigned int buffer[2];
	readData(buffer, d_lmerOffsets + lmerCount - 1, 1, sizeof(unsigned int));
	readData(buffer + 1, d_lmerValues + lmerCount - 1, 1, sizeof(unsigned int));
	*ecount = buffer[0] + buffer[1];

	logMessage(LOG_LVL_MSG, "debruijn vertex count:%d \ndebruijn edge count:%d",
			kmerCount, *ecount);

	allocateMemory((void**) d_ev, sizeof(EulerVertex) * (kmerCount));
	allocateMemory((void**) d_l, sizeof(unsigned int) * (*ecount));
	allocateMemory((void**) d_e, sizeof(unsigned int) * (*ecount));
	allocateMemory((void**) d_ee, sizeof(EulerEdge) * (*ecount));
	CheckCUDAError();
	hipMemset(*d_e, 0, sizeof(unsigned int) * (*ecount));
	hipMemset(*d_l, 0, sizeof(unsigned int) * (*ecount));
	CheckCUDAError();

	logMessage(LOG_LVL_DETAIL, "kernel: setupVertices");
	getOptimalLaunchConfiguration(kmerCount, &grid, &block);
	//cutilCheckError(cutStartTimer(timerGPU));
	setupVertices<<<grid,block>>>(d_kmerKeys,kmerCount,d_TK,d_TV,d_bucketSeed,bucketCount,*d_ev,d_lcount,d_lstart,d_ecount,d_estart);
	CheckCUDAError();

	///*DEBUG*/verifyleOffsets(d_lstart,d_lcount,d_estart,d_ecount,4*kmerCount,*ecount);

	getOptimalLaunchConfiguration(lmerCount, &grid, &block);
	//verifySetupEdges(d_lmerKeys,d_lmerValues,d_lmerOffsets,lmerCount, d_TK,d_TV,d_bucketSeed,bucketCount,*d_l,*d_e,*d_ee,d_lcount,d_lstart,d_ecount,d_estart,kmerCount,*ecount,validBitMask);
	logMessage(LOG_LVL_DETAIL,"kernel: setupEdges");
	setupEdges<<<grid,block>>>(d_lmerKeys,d_lmerValues,d_lmerOffsets,lmerCount, d_TK,d_TV,d_bucketSeed,bucketCount,*d_l,*d_e,*d_ee,d_lstart,d_estart,validBitMask);

	CheckCUDAError();

	//cutilCheckError(cutStopTimer(timerGPU));
	//logMessage(LOG_LVL_MSG,"CPU Time : %f",cutGetTimerValue(timerGPU));

	//constructDebruijnGold( d_idata, d_icount, kmerCount,kmerLength,totalVertices,validBitMask);
	//printDebruijnGraph(*d_ev, kmerCount, *d_l, *d_e, *d_ee, *ecount, k, 0); // may not need it

	//printDebruijnGraph(*d_ev,kmerCount,*d_l,*d_e,*d_ee,*ecount,k,1);
	//printData(*d_ev,*vcount);
	//printData(*d_ee,*ecount);

	//cutilCheckError(cutDeleteTimer(timerGPU));

	deallocateMemory(d_lmerOffsets);
	deallocateMemory(d_lcount);
	deallocateMemory(d_lstart);
	deallocateMemory(d_estart);
	deallocateMemory(d_ecount);

}

