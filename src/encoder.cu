#include "hip/hip_runtime.h"
#define NPY_NO_DEPRECATED_API NPY_1_7_API_VERSION
#define CUDPP_STATIC_LIB


// #include <cutil_inline.h>
#include "/Volumes/Macintosh HD/Developer/NVIDIA/CUDA-7.5/samples/common/inc/hip/hip_runtime_api.h" 
#include <stdio.h>      /* defines printf for tests */
#include <time.h>       /* defines time_t for timings in the test */
#include <math.h>

#ifdef linux
# include <endian.h>    /* attempt to define endianness */
#endif


#include "Kmer.h"
#include "Graph.h"
#include "cudpp.h"
#include "utils.h"
#include "common.h"
#include "encode_kernel.cu"

// idea ! have all the cuda code as macro to be used in cpu code as well cuda code

/*
one read per block
|R| threads
copy each Ri to shared mem
|lmers|=|R|
l-1 dummy enteris or l-1 thread stall/branch

max overhead 256 bytes per read.

1) use dummy enteris
2) stall extra threads.
*/
//A=65=41=0100-0001
//C=67=43=0100-0011
//T=84=54=0101-0100
//G=71=47=0100-1111
//0A0CT00G
//00013002
//0T0GA00C
//03020001
//__device__ __constant__ KEY_T lmerMask[] ={
//    0x0000000000000003, 0x000000000000000F, 0x000000000000003F, 0x00000000000000FF, // 0   1   2   3
//    0x00000000000003FF, 0x0000000000000FFF, 0x0000000000003FFF, 0x000000000000FFFF, // 4   5   6   7
//    0x000000000003FFFF, 0x00000000000FFFFF, 0x00000000003FFFFF, 0x0000000000FFFFFF, // 8   9   10  11
//    0x0000000003FFFFFF, 0x000000000FFFFFFF, 0x000000003FFFFFFF, 0x00000000FFFFFFFF, // 12  13  14  15
//    0x00000003FFFFFFFF, 0x0000000FFFFFFFFF, 0x0000003FFFFFFFFF, 0x000000FFFFFFFFFF, // 16  17  18  19
//    0x000003FFFFFFFFFF, 0x00000FFFFFFFFFFF, 0x00003FFFFFFFFFFF, 0x0000FFFFFFFFFFFF, // 20  21  22  23
//    0x0003FFFFFFFFFFFF, 0x000FFFFFFFFFFFFF, 0x003FFFFFFFFFFFFF, 0x00FFFFFFFFFFFFFF, // 24  25  26  27
//    0x03FFFFFFFFFFFFFF, 0x0FFFFFFFFFFFFFFF, 0x3FFFFFFFFFFFFFFF, 0xFFFFFFFFFFFFFFFF // 28  29  30  31
//};
//
//__device__ __constant__ unsigned char shifter[4] [4]=
//{
//		{0,0,0,0},
//		{1,4,16,64},
//		{2,8,32,128},
//		{3,12,48,192},
//};
//
//__device__ __constant__ char  codeF[]={0,0,0,1,3,0,0,2};
//__device__ __constant__ char  codeR[]={0,3,0,2,0,0,0,1};
//__global__ void encodeLmerDevice(char  * buffer,
//			//	const unsigned int buffSize,
//			//	const unsigned int readLength,
//				KEY_PTR lmers,
//				const unsigned int lmerLength
//				)
//{
//    printf("%s", buffer);
//	extern __shared__ char dnaRead[]; // MB: changed from 'read' to solve compile error
//	const unsigned int tid=threadIdx.x;
//	const unsigned int rOffset=(blockDim.x*blockDim.y*gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x) + (blockDim.x*threadIdx.y);
//	KEY_T lmer=0;
//
//	dnaRead[tid]=buffer[rOffset+tid];
//	__syncthreads();
//
//	for (unsigned int i = 0; i < 8; i++)    //calculate lmer
//	{
//	    lmer= (lmer<< 8) |	((KEY_T)(shifter[codeF[dnaRead[threadIdx.x+i*4]& 0x07]][3] |
//							shifter[codeF[dnaRead[threadIdx.x+i*4+1]& 0x07]][2] |
//							shifter[codeF[dnaRead[threadIdx.x+i*4+2]& 0x07]][1] |
//							codeF[dnaRead[threadIdx.x+i*4+3] & 0x07]) ) ;
//	}
//	lmer = (lmer >> ((32 - lmerLength) << 1)) & lmerMask[lmerLength-1];
//
//	lmers[rOffset+tid]=lmer;
//
//
//}
//__global__ void encodeLmerComplementDevice(	char  * buffer,
//				const unsigned int buffSize,
//				const unsigned int readLength,
//				KEY_PTR lmers,
//				const unsigned int lmerLength
//				){
//
//	extern __shared__ char dnaRead[];//have to fix it
//	const unsigned int tid=threadIdx.x;
//	const unsigned int rOffset=(blockDim.x*blockDim.y*gridDim.x*blockIdx.y) +(blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y);
//	KEY_T lmer=0;
//	KEY_T temp=0;
//
//	dnaRead[tid]=buffer[rOffset+tid];
//	__syncthreads();
//	dnaRead[tid]=codeR[dnaRead[tid] & 0x07];
//	__syncthreads();
//	for(unsigned int i =0; i< lmerLength; i++){
//		temp=((KEY_T)dnaRead[(tid+i)%blockDim.x]);
//		lmer = (temp<<(i<<1)) | lmer;
//	}
//	lmers[rOffset+tid]=lmer;
//
//}


//__global__ void computeKmerDevice( 	KEY_PTR lmers,
//				KEY_PTR pkmers,
//				KEY_PTR skmers,
//				KEY_T validBitMask
//			)
//{
//
//	const unsigned int tid=(blockDim.x*blockDim.y*gridDim.x*blockIdx.y) +(blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
//	KEY_T lmer;
//	//fetch lmer
//	lmer=lmers[tid];
//	//find prefix
//	pkmers[tid]=LMER_PREFIX(lmer,validBitMask);
//	//find suffix
//	skmers[tid] = LMER_SUFFIX(lmer,validBitMask);
//}

extern "C"
void encodeLmer(
		char * d_buffer,
		const unsigned int bufferSize,
		const unsigned int readLength,
		KEY_PTR d_lmers,
		const unsigned int lmerLength,
		const unsigned int entriesCount
		)
{
	dim3 grid, block;
	char * d_reads = NULL;

	hipError_t err = hipMalloc((void**) &d_reads, bufferSize);
    err = hipMemcpy(d_reads, d_buffer, bufferSize,hipMemcpyHostToDevice);
    if (err != 0)
        printf("hipMemcpy error encodeLmer: %d\n",err);
	getOptimalLaunchConfigCustomized(entriesCount,&grid,&block,readLength);

	encodeLmerDevice<<<grid,block,readLength+31>>>(d_reads,d_lmers,lmerLength);

	CheckCUDAError();
		

}
extern "C"
void encodeLmerComplement(
		char * d_buffer,
		const unsigned int bufferSize,
		const unsigned int readLength,
		KEY_PTR d_lmers,
		const unsigned int lmerLength,
		const unsigned int entriesCount
		)
{
	dim3 grid, block;
	char * d_reads = NULL;

	hipError_t err = hipMalloc((void**) &d_reads, bufferSize);
    err = hipMemcpy(d_reads, d_buffer, bufferSize,hipMemcpyHostToDevice);
    if (err != 0)
        printf("hipMemcpy error encodeLmerComplement: %d\n",err);
	getOptimalLaunchConfigCustomized(entriesCount,&grid,&block,readLength);		
	encodeLmerComplementDevice<<<grid,block,readLength>>>(d_reads,bufferSize,readLength,d_lmers,lmerLength);
	CheckCUDAError();
}

extern "C"
void computeKmer(	KEY_PTR d_lmers,
			KEY_PTR d_pkmers,
			KEY_PTR d_skmers,
			KEY_PTR h_lmers,
			KEY_PTR h_pkmers,
			KEY_PTR h_skmers,
			KEY_T validBitMask,
			const unsigned int readLength,
			const unsigned int entriesCount
		)
{
	dim3 grid, block;
	unsigned int ebSize = entriesCount * sizeof(KEY_T);
	getOptimalLaunchConfigCustomized(entriesCount,&grid,&block,readLength);
	computeKmerDevice<<<grid,block>>>(d_lmers,d_pkmers,d_skmers,validBitMask);

    hipError_t err1 = hipMemcpy(h_lmers, d_lmers, ebSize, hipMemcpyDeviceToHost);
    hipError_t err2 = hipMemcpy(h_pkmers, d_pkmers, ebSize,hipMemcpyDeviceToHost);
    hipError_t err3 = hipMemcpy(h_skmers, d_skmers, ebSize,hipMemcpyDeviceToHost);

    if ((err1 != 0) || (err2 != 0) || (err3 != 0))
        printf("err1 = %d err2 = %d err3 = %d\n",err1, err2, err3);

	CheckCUDAError();
			
}

//extern "C"
//void computeKmerComplement(	KEY_PTR d_lmers,
//			KEY_PTR d_pkmers,
//			KEY_PTR d_skmers,
//			KEY_PTR h_lmers,
//			KEY_PTR h_pkmers,
//			KEY_PTR h_skmers,
//			KEY_T validBitMask,
//			const unsigned int readLength,
//			const unsigned int entriesCount
//		)
//{
//	dim3 grid, block;
//	unsigned int ebSize = entriesCount * sizeof(KEY_T);
//	getOptimalLaunchConfigCustomized(entriesCount,&grid,&block,readLength);
//	computeKmerDevice<<<grid,block>>>(d_lmers,d_pkmers,d_skmers,validBitMask);
//
//    hipError_t err1 = hipMemcpy(h_lmers, d_lmers, ebSize, hipMemcpyDeviceToHost);
//    hipError_t err2 = hipMemcpy(h_pkmers, d_pkmers, ebSize,hipMemcpyDeviceToHost);
//    hipError_t err3 = hipMemcpy(h_skmers, d_skmers, ebSize,hipMemcpyDeviceToHost);
//
//    if ((err1 != 0) || (err2 != 0) || (err3 != 0))
//        printf("err1 = %d err2 = %d err3 = %d\n",err1, err2, err3);
//
//	CheckCUDAError();
//
//}
