#include "hip/hip_runtime.h"
#include "/Volumes/Macintosh HD/Developer/NVIDIA/CUDA-7.5/samples/common/inc/hip/hip_runtime_api.h" // lib above replaced w/this one at CUDA 5.0
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdarg.h>
#include "Graph.h"
#include "common.h"
 #include <execinfo.h>
#define DEFAULT_BLOCK_SIZE 512

#ifdef EULER_NDEBUG
#define DEBUG_UTIL_CPP(x)
#else
#define DEBUG_UTIL_CPP(x) x
#endif
#define DEBUG_CALL(x) DEBUG_UTIL_CPP(x)

#define LOG_ENABLED 
typedef struct MemNode {
	void * d_ptr;
	MemNode * next;
} MemNode;

/********** Globals *****/
MemNode * head;
int LOG_LEVEL = 0;
int blockSize = DEFAULT_BLOCK_SIZE;
FILE * logFile;

/* function for loggin purpose**/


extern "C" void logMessage(int logLevel, const char * format, ...) {

#if defined LOG_ENABLED
		if( logLevel<=LOG_LEVEL) {
			va_list ap;
			int r;
			va_start (ap, format);
			fprintf(stderr,"<!-- ");
			//printf ("P[%d]:",rank);
		r = vfprintf (stderr,format, ap);
		va_end (ap);
		fprintf(stderr," --> \n");
		//	fprintf(stderr,"\n");

	}
#endif
	}
extern "C" void logMessageNL(int logLevel, const char * format, ...) { //this is actually logMessage No LineFeed

#if defined LOG_ENABLED
		if( logLevel<=LOG_LEVEL) {
			va_list ap;
			int r;
			va_start (ap, format);
			fprintf(stderr,"<!-- ");
			//printf ("P[%d]:",rank);
		r = vfprintf (stderr,format, ap);
		fprintf(stderr," -->");
		va_end (ap);

	}
#endif
	}

void logCaller(){
	 void *buffer[6];
	 char **strings;

	 unsigned int nptrs = backtrace(buffer, 5);
	strings = backtrace_symbols(buffer, nptrs);
	if (strings == NULL) {
		perror("backtrace_symbols");

	}else {
		logMessage(LOG_LVL_DEBUG, "Caller:%s  , Callee:%s", strings[2],strings[1]);
		 free(strings);
	}


}
extern "C" void CheckCUDAError() {
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		logMessage(LOG_LVL_ERROR, "%s\n", hipGetErrorString(err));
	}else{
		logMessage(LOG_LVL_DETAIL, "CUDA call succeeded");
	}
}

extern "C" void initMemList() {
	head = (MemNode *) malloc(sizeof(MemNode));
	head->d_ptr = NULL;
	head->next = NULL;
}

extern "C" void addMemNode(void * d_ptr) {
	MemNode * tmp = (MemNode*) malloc(sizeof(MemNode));
	tmp->d_ptr = d_ptr;
	tmp->next = head->next;
	head->next = tmp;
}

extern "C" void freeMemList() {
	MemNode * tmp;
	while (head->next != NULL) {
		tmp = head->next;
		head->next = head->next->next;
		hipFree(tmp->d_ptr);
		free(tmp);
		CheckCUDAError();
	}
}
extern "C" void cleanupMemList() {
	freeMemList();
	free(head);
}
extern "C" void printData(unsigned int * d_buffer, int length) {

	unsigned int *h_buffer = (unsigned int *) malloc(length * sizeof(int));
	checkCudaErrors(
			hipMemcpy(h_buffer, d_buffer, length * sizeof(int),
					hipMemcpyDeviceToHost));
	for (int i = 0; i < length; i++) {
		logMessageNL(LOG_LVL_MSG, "@[%d] %u \n", i, h_buffer[i]);
	}
	logMessage(LOG_LVL_DETAIL, "");
	free(h_buffer);

}
/*
 extern "C"
 void printData(void * d_buffer,int length,int width){

 unsigned int *h_buffer= (unsigned int *)malloc(length* sizeof(int)*width);
 checkCudaErrors( hipMemcpy( h_buffer, d_buffer, length* sizeof(int)*width,hipMemcpyDeviceToHost) );
 for( int i=0;i<length;i++){
 for(int j=0;j< width;j++) {
 printf("[%d]:%u ",j,h_buffer[i*length+j]);
 }
 printf("\n");
 }
 printf("\n");
 free(h_buffer);

 }*/
/*
 extern "C"
 void getOptimalLaunchConfiguration(unsigned int threadCount,unsigned int * gridx,unsigned int * gridy,unsigned int * threads){

 *threads=32;
 *gridy=threadCount/(*threads);
 if(threadCount%(*threads) >0) (*gridy)++;
 (*gridx)=(*gridy)/65535+1;
 (*gridy)=(*gridy)%65535;
 }
 */

extern "C" void setBlockSize(int newSize) {
	blockSize = newSize;
}

extern "C" void getOptimalLaunchConfigCustomized(unsigned int threadCount,
		dim3 * grid, dim3 * block, unsigned int threadPerBlock) {

	*block = make_uint3(threadPerBlock, 1, 1);
	*grid = make_uint3(1, 1, 1);

	/*grid->y=threadCount/(block->x);
	 if(threadCount%(block->x) >0) grid->y++;
	 grid->x=grid->y / 65535 +1;
	 grid->y=grid->y % 65535;
	 grid->z=1;*/
	if (threadCount > block->x) {
		grid->y = threadCount / (block->x);
		if (threadCount % (block->x) > 0)
			grid->y++;
		grid->x = grid->y / 65535 + 1;
		grid->y = (grid->y > 65535 ) ? 65535 : grid->y;
		grid->z = 1;
	}
}
extern "C" void getOptimalLaunchConfiguration(unsigned int threadCount,
		dim3 * grid, dim3 * block) {
	getOptimalLaunchConfigCustomized(threadCount, grid, block, blockSize);
}

extern "C" void readData(void * h_out, void * d_in, int length, int width) {
	checkCudaErrors(
			hipMemcpy(h_out, d_in, length * width, hipMemcpyDeviceToHost));
}

extern "C" void allocateMemory(void ** d_buffer, unsigned int memSize) {
	size_t  free, total;
	DEBUG_CALL(logCaller());
	hipMemGetInfo(&free, &total);
	logMessage(LOG_LVL_DEBUG, "\t\t\tMemory Requested %d bytes", memSize);
	logMessage(
			LOG_LVL_DEBUG,
			"\t\t\tMemory Status Before Alloc :: total:[%u]\t used:[%u]\t free:[%u]",
			total, total - free, free);
	if (total - free != 0 && free < memSize) {
		freeMemList();
	}
	checkCudaErrors( hipMalloc((void**) d_buffer, memSize));
	;
	CheckCUDAError();
	logMessage(LOG_LVL_DEBUG, "\t\t\tmemory address %u", *d_buffer);
	hipMemset(*d_buffer, 0, memSize);
	hipMemGetInfo(&free, &total);
	logMessage(
			LOG_LVL_DEBUG,
			"\t\t\tMemory Status After Alloc :: total:[%u]\t used:[%u]\t free:[%u]",
			total, total - free, free);
}

extern "C" void deallocateMemory(void * d_buffer) {

	size_t freeBefore, total, freeAfter;
	DEBUG_CALL(logCaller());
	logMessage(LOG_LVL_DEBUG, "\t\t\tReleasing Memory");

	hipMemGetInfo(&freeBefore, &total);
	logMessage(
			LOG_LVL_DEBUG,
			"\t\t\tMemory Status Before Releasing :: total:[%u]\t used:[%u]\t free:[%u]",
			total, total - freeBefore, freeBefore);
	//hipFree(d_buffer) ;//
	checkCudaErrors(hipFree(d_buffer));
	//addMemNode(d_buffer);
	logMessage(LOG_LVL_DEBUG, "\t\t\t adding memory to junk list %u\n",
			d_buffer);
	CheckCUDAError();
	hipMemGetInfo(&freeAfter, &total);
	logMessage(
			LOG_LVL_DEBUG,
			"\t\t\tMemory Status After Releasing :: total:[%u]\t used:[%u]\t free:[%u]\t",
			total, total - freeAfter, freeAfter);
	logMessage(LOG_LVL_DEBUG, "\t\t\tMemory Freed %d bytes\n",
			freeAfter - freeBefore);

}
char _translate(int i) {
	if (i == 0)
		return 'A';
	if (i == 1)
		return 'C';
	if (i == 2)
		return 'G';
	if (i == 3)
		return 'T';
	return '.';
}
void _getString(char * kmer, int length, unsigned int value) {

	unsigned int currentValue = value;
	for (int i = 1; i <= length; i++) {
		kmer[length - i] = _translate(currentValue % 4);
		currentValue = currentValue / 4;
	}
}

extern "C" void printDebruijnGraphLongFmt(EulerVertex * d_ev, int vertexCount,
		unsigned int * d_l, unsigned int * d_e, EulerEdge * d_ee, int edgeCount,
		unsigned int kmerLength) {
	EulerVertex *h_ev = (EulerVertex *) malloc(
			vertexCount * sizeof(EulerVertex));
	EulerEdge *h_ee = (EulerEdge *) malloc(edgeCount * sizeof(EulerEdge));
	unsigned int * h_l = (unsigned int *) malloc(
			edgeCount * sizeof(unsigned int));
	unsigned int * h_e = (unsigned int *) malloc(
			edgeCount * sizeof(unsigned int));
	checkCudaErrors(
			hipMemcpy(h_ev, d_ev, vertexCount * sizeof(EulerVertex),
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_ee, d_ee, edgeCount * sizeof(EulerEdge),
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_e, d_e, edgeCount * sizeof(unsigned int),
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_l, d_l, edgeCount * sizeof(unsigned int),
					hipMemcpyDeviceToHost));

	for (int i = 0; i < vertexCount; i++) {
		logMessage(LOG_LVL_DEBUG,
				"$[%d]:vid[%lu], ep[%u], eend[%u],  lp[%u], lend[%u]", i,
				(unsigned long) h_ev[i].vid, h_ev[i].ep, h_ev[i].ecount,
				h_ev[i].lp, h_ev[i].lcount);
		logMessageNL(LOG_LVL_DEBUG, "$e: [");
		for (unsigned int j = 0; j < h_ev[i].ecount; j++) {
			logMessageNL(LOG_LVL_DEBUG, " %d", h_e[h_ev[i].ep + j]);
		}
		logMessage(LOG_LVL_DEBUG, "]");
		logMessageNL(LOG_LVL_DEBUG, "$l: [");
		for (unsigned int j = 0; j < h_ev[i].lcount; j++) {
			logMessageNL(LOG_LVL_DEBUG, " %d", h_l[h_ev[i].lp + j]);
		}
		logMessage(LOG_LVL_DEBUG, "]");

	}

	logMessageNL(LOG_LVL_DEBUG, "$e: [");
	for (int i = 0; i < edgeCount; i++) {
		logMessageNL(LOG_LVL_DEBUG, " %u", h_e[i]);
	}
	logMessage(LOG_LVL_DEBUG, "]");
	logMessageNL(LOG_LVL_DEBUG, "$l: [");
	for (int i = 0; i < edgeCount; i++) {
		logMessageNL(LOG_LVL_DEBUG, " %u", h_l[i]);
	}
	logMessage(LOG_LVL_DEBUG, "]");

	logMessage(LOG_LVL_DEBUG, "$edges...");
	for (int i = 0; i < edgeCount; i++) {
		logMessage(LOG_LVL_DEBUG, "$[%d]: eid[%u], v1[%u], v2[%u], s[%u]\n", i,
				h_ee[i].eid, h_ee[i].v1, h_ee[i].v2, h_ee[i].s);
	}

	free(h_l);
	free(h_e);
	free(h_ev);
	free(h_ee);
}
extern "C" void printDebruijnGraphVizFmt(EulerVertex * d_ev, int vertexCount,
		unsigned int * d_l, unsigned int * d_e, EulerEdge * d_ee, int edgeCount,
		unsigned int kmerLength) {

	EulerVertex *h_ev = (EulerVertex *) malloc(
			vertexCount * sizeof(EulerVertex));
	EulerEdge *h_ee = (EulerEdge *) malloc(edgeCount * sizeof(EulerEdge));
	unsigned int * h_l = (unsigned int *) malloc(
			edgeCount * sizeof(unsigned int));
	unsigned int * h_e = (unsigned int *) malloc(
			edgeCount * sizeof(unsigned int));
	checkCudaErrors(
			hipMemcpy(h_ev, d_ev, vertexCount * sizeof(EulerVertex),
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_ee, d_ee, edgeCount * sizeof(EulerEdge),
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_e, d_e, edgeCount * sizeof(unsigned int),
					hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy(h_l, d_l, edgeCount * sizeof(unsigned int),
					hipMemcpyDeviceToHost));

	char * v1 = (char *) malloc(kmerLength * sizeof(char));
	v1[kmerLength - 1] = '\0';

	char * v2 = (char *) malloc(kmerLength * sizeof(char));
	v2[kmerLength - 1] = '\0';

	logMessage(LOG_LVL_DEBUG, "$digraph G{");
	for (int i = 0; i < vertexCount; i++) {
		//_getString(v2,kmerLength-1,h_ev[i].vid);
		_getString(v1, kmerLength - 1, h_ev[i].vid);
		for (unsigned int j = 0; j < h_ev[i].ecount; j++) {
			//_getString(v1,kmerLength-1,h_ev[h_ee[h_e[h_ev[i].ep+j]].v1].vid);
			_getString(v2, kmerLength - 1,
					h_ev[h_ee[h_l[h_ev[i].lp + j]].v2].vid);
			//printf("$\t%s -> %s [ label= %s]\n",v1,v2,v2+kmerLength-2);
			logMessageNL(LOG_LVL_DEBUG, "$\t%s -> %s [ label= %s]\n", v1, v2,
					v2 + kmerLength - 2);
		}
	}
	logMessage(LOG_LVL_DEBUG, "$}");

	free(h_l);
	free(h_e);
	free(h_ev);
	free(h_ee);
	free(v1);
	free(v2);
}
extern "C" void printDebruijnGraph(EulerVertex * d_ev, unsigned int vertexCount,
		unsigned int * d_l, unsigned int * d_e, EulerEdge * d_ee,
		unsigned int edgeCount, unsigned int kmerLength, int format) {
	if (LOG_LEVEL >= LOG_LVL_DEBUG) {
		switch (format) {
		case 0:
			printDebruijnGraphLongFmt(d_ev, vertexCount, d_l, d_e, d_ee,
					edgeCount, kmerLength);
			break;
		case 1:
			printDebruijnGraphVizFmt(d_ev, vertexCount, d_l, d_e, d_ee,
					edgeCount, kmerLength);
			break;
		}
	}
}
extern "C" void printDeviceInfo(int argc, char** argv) {

	int devID;
	hipDeviceProp_t deviceProp;

	// get number of SMs on this GPU
	checkCudaErrors(hipGetDevice(&devID));
	checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));

	if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
		// printf("[%s] - there is no device supporting CUDA.\n", sSDKsample);
		hipDeviceReset();
//		cutilExit(argc, argv);
	} else {
		printf("#> Device %d: \"%s\"\n", devID, deviceProp.name);
		printf("#> SM Capability %d.%d detected:\n", deviceProp.major,
				deviceProp.minor);
	}
	printf("#Device %d: \"%s\"\n", 0, deviceProp.name);
	printf("#  CUDA Capability Major revision number:         %d\n",
			deviceProp.major);
	printf("#  CUDA Capability Minor revision number:         %d\n",
			deviceProp.minor);
	printf("#  Total amount of global memory:                 %lu bytes\n",
			(unsigned long) deviceProp.totalGlobalMem);
	// #if CUDART_VERSION >= 2000
	printf("#  Number of multiprocessors:                     %d\n",
			deviceProp.multiProcessorCount);
	printf("#  Number of cores:                               %d\n",
			8 * deviceProp.multiProcessorCount);
	// #endif
	printf("#  Total amount of constant memory:               %lu bytes\n",
			(unsigned long) deviceProp.totalConstMem);
	printf("#  Total amount of shared memory per block:       %lu bytes\n",
			(unsigned long) deviceProp.sharedMemPerBlock);
	printf("#  Total number of registers available per block: %d\n",
			deviceProp.regsPerBlock);
	printf("#  Warp size:                                     %d\n",
			deviceProp.warpSize);
	printf("#  Maximum number of threads per block:           %d\n",
			deviceProp.maxThreadsPerBlock);
	printf("#  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
			deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
			deviceProp.maxThreadsDim[2]);
	printf("#  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
			deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
			deviceProp.maxGridSize[2]);
	printf("#  Maximum memory pitch:                          %lu bytes\n",
			(unsigned long) deviceProp.memPitch);
	printf("#  Texture alignment:                             %lu bytes\n",
			(unsigned long) deviceProp.textureAlignment);
	printf("#  Clock rate:                                    %.2f GHz\n",
			deviceProp.clockRate * 1e-6f);
	// #if CUDART_VERSION >= 2000
	printf("#  Concurrent copy and execution:                 %s\n",
			deviceProp.deviceOverlap ? "Yes" : "No");
	//  #endif
	//  #if CUDART_VERSION >= 2020
	printf("#  Run time limit on kernels:                     %s\n",
			deviceProp.kernelExecTimeoutEnabled ? "Yes" : "No");
	printf("#  Integrated:                                    %s\n",
			deviceProp.integrated ? "Yes" : "No");
	printf("#  Support host page-locked memory mapping:       %s\n",
			deviceProp.canMapHostMemory ? "Yes" : "No");
	printf(
			"#  Compute mode:                                  %s\n",
			deviceProp.computeMode == hipComputeModeDefault ?
					"Default (multiple host threads can use this device simultaneously)" :
			deviceProp.computeMode == hipComputeModeExclusive ?
					"Exclusive (only one host thread at a time can use this device)" :
			deviceProp.computeMode == hipComputeModeProhibited ?
					"Prohibited (no host thread can use this device)" :
					"Unknown");
	// #endif
}

extern "C"
void initDevice() {
	checkCudaErrors(hipDeviceReset());

	checkCudaErrors(hipSetDevice(gpuGetMaxGflopsDeviceId() ));
	checkCudaErrors(hipSetDeviceFlags(hipDeviceMapHost));
	CheckCUDAError();
}

extern "C"
void resetDevice(){

}
