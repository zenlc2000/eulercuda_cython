#include "hip/hip_runtime.h"




/*
 *  This kernel works on each l-mer ,counting edges of the graph.
 */
__global__ void debruijnCount(
    KEY_PTR lmerKeys, /* lmer keys	*/
    VALUE_PTR lmerValues, /* lmer frequency */
    unsigned int lmerCount, /* total lmers */
    KEY_PTR TK, /* Keys' pointer for Hash table*/
    VALUE_PTR TV, /* Value pointer for Hash table*/
    unsigned int * bucketSeed, /* bucketSize: size of each bucket (it should be renamed to bucketSize)*/
    unsigned int bucketCount, /* total buckets */
    unsigned int * lcount, /* leaving edge count array : OUT */
    unsigned int * ecount, /* entering edge count array: OUT */
    KEY_T validBitMask /* bit mask for K length encoded bits*/
    )
{

	unsigned int tid = (blockDim.x * blockDim.y * gridDim.x * blockIdx.y)
			+ (blockDim.x * blockDim.y * blockIdx.x)
			+ (blockDim.x * threadIdx.y) + threadIdx.x;
	if (tid < lmerCount)
	{
		KEY_T lmer = lmerKeys[tid];
		VALUE_T lmerValue = lmerValues[tid];
		KEY_T prefix = (lmer & (validBitMask << 2)) >> 2;
		KEY_T suffix = (lmer & validBitMask);

		KEY_T lomask = 3;
		VALUE_T prefixIndex = getHashValue(prefix, TK, TV, bucketSeed,
				bucketCount);
		VALUE_T suffixIndex = getHashValue(suffix, TK, TV, bucketSeed,
				bucketCount);
		KEY_T transitionTo = (lmer & lomask);
		KEY_T transitionFrom = ((lmer >> __popcll(validBitMask)) & lomask);
		//atomicAdd(lcount+(prefixIndex<<2 )+transition,lmerValue);
		//atomicAdd(ecount+(suffixIndex<<2)+transition,lmerValue);
		lcount[(prefixIndex << 2) + transitionTo] = lmerValue;
		ecount[(suffixIndex << 2) + transitionFrom] = lmerValue;
	}
}

/*
 *  This kernel works on a k-mer (l-1mer) which are vertices of the graph.
 */

__global__ void setupVertices(KEY_PTR kmerKeys, unsigned int kmerCount,
		KEY_PTR TK, VALUE_PTR TV, unsigned int * bucketSeed,
		unsigned int bucketCount, EulerVertex * ev, unsigned int * lcount,
		unsigned int * loffset, unsigned int * ecount, unsigned int * eoffset)
{
	unsigned int tid = (blockDim.x * blockDim.y * gridDim.x * blockIdx.y)
			+ (blockDim.x * blockDim.y * blockIdx.x)
			+ (blockDim.x * threadIdx.y) + threadIdx.x;
	if (tid < kmerCount)
	{
		KEY_T key = kmerKeys[tid];
		VALUE_T index = getHashValue(key, TK, TV, bucketSeed, bucketCount);
		;
		ev[index].vid = key;
		ev[index].lp = loffset[(index << 2)];
		ev[index].lcount = lcount[(index << 2)] + lcount[(index << 2) + 1]
				+ lcount[(index << 2) + 2] + lcount[(index << 2) + 3];
		ev[index].ep = eoffset[(index << 2)];
		ev[index].ecount = ecount[(index << 2)] + ecount[(index << 2) + 1]
				+ ecount[(index << 2) + 2] + ecount[(index << 2) + 3];
	}
}

/*
 *  This kernel works on an l-mer, which represents an edge
 *  in the debruijn Graph.
 */
__global__ void setupEdges( KEY_PTR  lmerKeys,  VALUE_PTR  lmerValues,
		 unsigned int *  lmerOffsets, const unsigned int lmerCount,
		 KEY_PTR  TK, VALUE_PTR  TV, unsigned int *  bucketSeed,
		const unsigned int bucketCount, unsigned int *  l,
		 unsigned int *  e, EulerEdge *  ee,
		 unsigned int *  loffsets, unsigned int *  eoffsets,
		const KEY_T validBitMask)
{

	unsigned int tid = (blockDim.x * blockDim.y * gridDim.x * blockIdx.y)
			+ (blockDim.x * blockDim.y * blockIdx.x)
			+ (blockDim.x * threadIdx.y) + threadIdx.x;
	if (tid < lmerCount)
	{
		KEY_T lmer = lmerKeys[tid];
		VALUE_T lmerValue = lmerValues[tid];
		KEY_T prefix = (lmer & (validBitMask << 2)) >> 2;
		KEY_T suffix = (lmer & validBitMask);
		KEY_T lomask = 3;
		//prefix and suffix index must be less than kmer count
		VALUE_T prefixIndex = getHashValue(prefix, TK, TV, bucketSeed,
				bucketCount);
		VALUE_T suffixIndex = getHashValue(suffix, TK, TV, bucketSeed,
				bucketCount);
		KEY_T transitionTo = (lmer & lomask);
		KEY_T transitionFrom = ((lmer >> __popcll(validBitMask)) & lomask);
		unsigned int loffset = loffsets[(prefixIndex << 2) + transitionTo];
		unsigned int eoffset = eoffsets[(suffixIndex << 2) + transitionFrom];

		unsigned int lmerOffset = lmerOffsets[tid];
		for (unsigned int i = 0; i < lmerValue; i++)
		{

			ee[lmerOffset].eid =lmerOffset;
			ee[lmerOffset].v1 = prefixIndex;
			ee[lmerOffset].v2 = suffixIndex;
			// lmerOffset;
			ee[lmerOffset].s = lmerValues[lmerCount - 1]
					+ lmerOffsets[lmerCount - 1];

			l[loffset] = lmerOffset;
			e[eoffset] = lmerOffset;
			loffset++;
			eoffset++;
			lmerOffset++;
		}
	}
}