#include "hip/hip_runtime.h"
#define CUDPP_STATIC_LIB



//#include <cutil_inline.h>
#include "/Volumes/Macintosh HD/Developer/NVIDIA/CUDA-7.5/samples/common/inc/hip/hip_runtime_api.h" // lib above replaced w/this one at CUDA 5.0
#include "/Volumes/Macintosh HD/Developer/NVIDIA/CUDA-7.5/samples/common/inc/helper_timer.h"
#include "/Volumes/Macintosh HD/Developer/NVIDIA/CUDA-7.5/samples/common/inc/helper_functions.h"
#include "cudpp.h"
#include "utils.h"
#include "gpuhash_device2.h"
#include <time.h>


#ifdef EULER_NDEBUG
#define DEBUG_GPUHASH2_CU(x)
#else
#define DEBUG_GPUHASH2_CU(x) x
#endif
#define DEBUG_CALL(x) DEBUG_GPUHASH2_CU(x)


__global__ void phase12(	KEY_PTR  keys,
			unsigned int * offset, 
			unsigned int length,
			unsigned int* count,
			unsigned int bucketCount){

	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	if(tid<length){
		KEY_T key=keys[tid];
		unsigned int bucket=HASH_H(key,bucketCount);
		offset[tid]=atomicInc (count+bucket,MAX_INT);
		
	}
}
__global__ void copyToBucket2(	KEY_PTR keys,
				VALUE_PTR values,
				unsigned int * offset,
				unsigned int length,
				unsigned int* start,
				unsigned int bucketCount,
				KEY_PTR  bufferK,
				VALUE_PTR bufferV){

	unsigned tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;;

	if(tid<length){
		KEY_T key =keys[tid];
		unsigned int bucket=HASH_H(key,bucketCount);
		VALUE_T value=values[tid];
		unsigned int index=start[bucket]+offset[tid];
		//index=(index * BUCKET_ITEM_SIZE);
		bufferK[index]=key;
		bufferV[index]=value;
		//*(BUFFER_ITEM_KEY_PTR(buffer,index))=key;
		//*(BUFFER_ITEM_VALUE_PTR(buffer,index))=value;
	}
}

__global__ void phase22(KEY_PTR   bufferK,VALUE_PTR bufferV,
					unsigned int * start, unsigned int * count,
					unsigned int * bucketSeed,unsigned int bucketCount,
					KEY_PTR TK,VALUE_PTR TV,unsigned int * randomSeed,
					unsigned int seedCount){



		__shared__ unsigned int L[L2_ITEM_COUNT*3];
		__shared__ unsigned int pending;

		unsigned int blockCount=count[blockIdx.x];
		unsigned int key=bufferK[ ((start[blockIdx.x]+threadIdx.x)%blockCount)];
		unsigned int g[3];
		int tIdx=-1;
		bool unplaced=true;

		unsigned int iterations=0;
		unsigned int seedIdx=0;
		unsigned int seed=1;

		unsigned int id=threadIdx.x % blockCount;

		while ((unplaced) && seedIdx<seedCount){
			seed=*(randomSeed+blockIdx.x*seedCount+seedIdx);
			g[0]=HASH_G1(key,seed);
			g[1]=HASH_G2(key,seed);
			g[2]=HASH_G3(key,seed);
			tIdx=-1;
			do{
				atomicExch(&pending,0);
				__syncthreads();
				if(unplaced ) {
					tIdx= (tIdx+1)%3;
					atomicExch(L+tIdx*L2_ITEM_COUNT + g[tIdx],id);
				}
				__syncthreads();
				unplaced =(L[tIdx*L2_ITEM_COUNT + g[tIdx]] != id );
				if(unplaced )
					atomicExch(&pending,1);
				__syncthreads();
				iterations++;
			}while(pending && iterations<MAX_ITERATIONS);
			seedIdx++;
			__syncthreads();
		}
		__syncthreads();
		if(seedIdx>=seedCount){
					if (threadIdx.x==0) *(bucketSeed+blockIdx.x)=MAX_INT;
		}else {
			if (threadIdx.x==0) bucketSeed[blockIdx.x]=seed;
			g[0]=HASH_G1(key,seed);
			g[1]=HASH_G2(key,seed);
			g[2]=HASH_G3(key,seed);

			if(threadIdx.x<blockCount){
				TK[blockIdx.x * BLOCK_SIZE +tIdx*L2_ITEM_COUNT+g[tIdx]]=key;
				TV[blockIdx.x * BLOCK_SIZE +tIdx*L2_ITEM_COUNT+g[tIdx]]=bufferV[(start[blockIdx.x]+threadIdx.x)];
			}

		}

	}
unsigned int host_phase2(KEY_PTR   bufferK,VALUE_PTR bufferV,
		unsigned int * start, unsigned int * count,
		KEY_PTR TK,VALUE_PTR TV,
		unsigned int seedCount, unsigned int blockIdxx){

	unsigned int blockCount=count[blockIdxx];
	unsigned int blockStart=start[blockIdxx];
	unsigned int iterations=0;
	bool unplaced=true;
	unsigned int seed;
	unsigned int g[3];
	bool inserted=false;

	while(iterations<seedCount && unplaced){
		seed=rand();
		for(unsigned int j=0;j<BLOCK_SIZE;j++){	//set every entry of table to NULL
				TK[blockIdxx*BLOCK_SIZE+j]=MAX_INT;
				TV[blockIdxx*BLOCK_SIZE+j]=MAX_INT;
			}
		for(unsigned int i=0;i<blockCount;i++){
			KEY_T key=bufferK[blockStart+i];
			VALUE_T value=bufferV[blockStart+i];
			//insert
			inserted=false;
			unsigned int tries=0;
			unsigned int gidx=0;
			KEY_T tempKey;
			VALUE_T tempValue;
			while(!inserted && tries<MAX_ITERATIONS){
				g[0]=HASH_G1(key,seed);
				g[1]=HASH_G2(key,seed);
				g[2]=HASH_G3(key,seed);

				tempKey=TK[blockIdxx*BLOCK_SIZE+L2_ITEM_COUNT*gidx+g[gidx]];
				tempValue=TV[blockIdxx*BLOCK_SIZE+L2_ITEM_COUNT*gidx+g[gidx]];
				TK[blockIdxx*BLOCK_SIZE+L2_ITEM_COUNT*gidx+g[gidx]]=key;
				TV[blockIdxx*BLOCK_SIZE+L2_ITEM_COUNT*gidx+g[gidx]]=value;
				if(tempKey==MAX_INT && tempKey==MAX_INT){
						//empty

					inserted=true;
				}else {
					key=tempKey;
					value=tempValue;
					gidx=((gidx+1)%3);
				}
				tries ++;
			}
			if(!inserted) break;
		}
		if(inserted) unplaced=false;
		iterations++;
	}
	if(iterations>=seedCount)
		return MAX_INT;
	else return seed;

}
void phase2HostLaunch(KEY_PTR   bufferK,VALUE_PTR bufferV,
					unsigned int * start, unsigned int * count,
					unsigned int * bucketSeed,unsigned int bucketCount,
					KEY_PTR TK,VALUE_PTR TV,
					unsigned int seedCount,unsigned int length){


	KEY_PTR h_bufferK=(KEY_PTR)malloc(KEY_SIZE*length);
	VALUE_PTR h_bufferV=(VALUE_PTR)malloc(VALUE_SIZE*length);
	KEY_PTR h_TK=(KEY_PTR)malloc(KEY_SIZE*BLOCK_SIZE*bucketCount);
	VALUE_PTR h_TV=(VALUE_PTR)malloc(VALUE_SIZE*BLOCK_SIZE*bucketCount);

	unsigned int * h_start=(unsigned int*)malloc(sizeof(unsigned int)* bucketCount);
	unsigned int * h_count=(unsigned int*)malloc(sizeof(unsigned int)* bucketCount);
	unsigned int * h_bucketSeed=(unsigned int*)malloc(sizeof(unsigned int)* bucketCount);


	checkCudaErrors( hipMemcpy(h_bufferK, bufferK, length * (KEY_SIZE), hipMemcpyDeviceToHost));
	checkCudaErrors( hipMemcpy(h_bufferV, bufferV, length * (VALUE_SIZE), hipMemcpyDeviceToHost));
	checkCudaErrors( hipMemcpy(h_start, start,sizeof(unsigned int)* bucketCount, hipMemcpyDeviceToHost));
	checkCudaErrors( hipMemcpy(h_count, count, sizeof(unsigned int)* bucketCount, hipMemcpyDeviceToHost));

	for(unsigned int blockIdx=0;blockIdx<bucketCount;blockIdx++){

		h_bucketSeed[blockIdx]=host_phase2(h_bufferK,h_bufferV,h_start,h_count,h_TK,h_TV,seedCount,blockIdx);
	}

	checkCudaErrors( hipMemcpy( bucketSeed, h_bucketSeed, sizeof(unsigned int)* bucketCount,hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy( TK, h_TK, KEY_SIZE*BLOCK_SIZE*bucketCount,hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy( TV, h_TV, VALUE_SIZE*BLOCK_SIZE*bucketCount,hipMemcpyHostToDevice) );


	free(h_bufferK);
	free(h_bufferV);
	free(h_TK);
	free(h_TV);
	free(h_start);
	free(h_count);
	free(h_bucketSeed);

}
void verifyBuffer2(KEY_PTR d_keys,VALUE_PTR d_values,unsigned int length, KEY_PTR d_bufferK, VALUE_PTR d_bufferV, unsigned int * d_bucketOffset,unsigned int * d_bucketSize, unsigned int bucketCount){

	KEY_PTR h_keys;
	VALUE_PTR h_values;
	KEY_PTR	h_bufferK;
	VALUE_PTR h_bufferV;
	unsigned int * h_bucketOffset;
	unsigned int * h_bucketSize;
	
	
	h_keys=(KEY_PTR) malloc(length * KEY_SIZE);
	h_values=(VALUE_PTR) malloc( length * VALUE_SIZE);
	h_bufferK= (KEY_PTR) malloc ( length *KEY_SIZE);
	h_bufferV= (VALUE_PTR) malloc( length * VALUE_SIZE);
	h_bucketOffset =(unsigned int *) malloc( bucketCount * sizeof(unsigned int));
	h_bucketSize = (unsigned int * ) malloc ( bucketCount * sizeof(unsigned int));	
	
	checkCudaErrors( hipMemcpy(h_keys, d_keys, length * (KEY_SIZE), hipMemcpyDeviceToHost));
	checkCudaErrors( hipMemcpy(h_values, d_values, length * (VALUE_SIZE), hipMemcpyDeviceToHost));
	checkCudaErrors( hipMemcpy(h_bufferK, d_bufferK, length * (KEY_SIZE), hipMemcpyDeviceToHost));
	checkCudaErrors( hipMemcpy(h_bufferV, d_bufferV, length * (VALUE_SIZE), hipMemcpyDeviceToHost));
	checkCudaErrors( hipMemcpy(h_bucketOffset, d_bucketOffset, bucketCount* (sizeof(unsigned int)), hipMemcpyDeviceToHost));
	checkCudaErrors( hipMemcpy(h_bucketSize, d_bucketSize, bucketCount * (sizeof(unsigned int)), hipMemcpyDeviceToHost));

	unsigned int found=0;
	unsigned int notfound=0;
	unsigned int correctValue=0;
	unsigned int incorrectValue=0;

	for (unsigned int i=0; i<length ; i++){
		unsigned int j=0;
		unsigned int bucket= HASH_H(h_keys[i],bucketCount);
		unsigned int offset=h_bucketOffset[bucket];
		unsigned int size = h_bucketSize[bucket];
	
		while( j<size && h_bufferK[offset+j]!=h_keys[i]) j++;
		if( j<size) {
			found ++;
			if( h_values[i]== h_bufferV[offset+j]) {
				correctValue++;
			}else {
				incorrectValue++;
			}
		}
		else {
			notfound++;
			incorrectValue++;
		}
		
	}
	printf("found:[%u], notfound:[%u] , correct:[%u] ,incorrect:[%u]\n",found,notfound,correctValue,incorrectValue);
	free(h_keys);
	free(h_values);
	free(h_bufferK);
	free(h_bufferV);
	free(h_bucketOffset);
	free(h_bucketSize);
}
void verifyHashTable2(KEY_PTR  d_keys, VALUE_PTR d_values, unsigned int length, KEY_PTR  d_TK,VALUE_PTR d_TV ,unsigned int tableLength, unsigned int * d_bucketSize, unsigned int bucketCount){

	KEY_PTR		 h_keys;
	VALUE_PTR	 h_values;
	KEY_PTR		 h_TK;
	VALUE_PTR	 h_TV;
	unsigned int * 	 h_bucketSize;
	
	unsigned int *	 bCount;
	unsigned int b;
	
	h_keys=(KEY_PTR) malloc(length * (KEY_SIZE));
	h_values=(VALUE_PTR) malloc(length * (VALUE_SIZE));
	h_TK= (KEY_PTR ) malloc( BUCKET_KEY_SIZE* bucketCount);
	h_TV= (VALUE_PTR ) malloc( BUCKET_VALUE_SIZE* bucketCount);
	h_bucketSize= (unsigned int *) malloc ( bucketCount * sizeof(unsigned int));
	bCount= (unsigned int * ) malloc( bucketCount *sizeof(unsigned int));
	
	checkCudaErrors( hipMemcpy(h_keys, d_keys, length * (KEY_SIZE), hipMemcpyDeviceToHost));
	checkCudaErrors( hipMemcpy(h_values, d_values, length * (VALUE_SIZE), hipMemcpyDeviceToHost) );
	checkCudaErrors( hipMemcpy(h_TK, d_TK,  BUCKET_KEY_SIZE*bucketCount, hipMemcpyDeviceToHost) );
	checkCudaErrors( hipMemcpy(h_TV, d_TV,  BUCKET_VALUE_SIZE*bucketCount, hipMemcpyDeviceToHost) );
	checkCudaErrors( hipMemcpy(h_bucketSize, d_bucketSize, bucketCount * sizeof(unsigned int), hipMemcpyDeviceToHost) );
	
	CheckCUDAError();
	/*for(int j=0; j<bucketCount; j++){
		printf(" [%u]:%u ,",h_bucketSeed[j],j);
		}
		printf("\n");*/

	unsigned int correct=0;
	unsigned int incorrect=0;
	
	memset(bCount,0,bucketCount * sizeof(unsigned int) );
	for (int i=0;i <length; i++){
		if( host_getHashValue2(h_keys[i],h_TK,h_TV,h_bucketSize,bucketCount,&b) == h_values[i]) {
			correct++;
			bCount[b]++;
		}
		else incorrect++;
	}
	printf("total verified %u, incorrect %u\n",correct,incorrect);
	
	//for(int  i =0; i<bucketCount; i++){ printf(" Count Bucket-%d=%u\n",i,bCount[i]);}
	free(h_keys);
	free(h_values);
	free(h_TK);
	free(h_TV);
	free(h_bucketSize);
	free(bCount);
	
}
extern "C"
void createHashTable2(KEY_PTR d_keys,VALUE_PTR d_values, unsigned int length, KEY_PTR *  d_TK,VALUE_PTR * d_TV,unsigned int * tableLength, unsigned int ** d_bucketSeed,unsigned int * bucketCount){


	unsigned int * d_count;
	unsigned int * d_offset;	
	unsigned int * d_start;
	
	KEY_PTR d_bufferK;
	VALUE_PTR d_bufferV;
	unsigned int * d_randomSeed;
	unsigned int * h_randomSeed;

//	unsigned int timer = 0;
//	cutilCheckError(cutCreateTimer(&timer));
	StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);
	srand ( time(NULL) );
	*bucketCount=(length /409)+1; //ceil
	unsigned int dataSize=length*sizeof(unsigned int);
	unsigned int bucketDataSize=*bucketCount*sizeof(unsigned int);

	checkCudaErrors( hipMalloc( (void**) &d_offset, dataSize));
	//allocate count 
	checkCudaErrors( hipMalloc( (void**) &d_count, bucketDataSize));
	
	
	//initialize offset to zero
	checkCudaErrors( hipMemset(d_offset,0,dataSize));
	//initialize count to zero
	checkCudaErrors( hipMemset(d_count,0,bucketDataSize));
	
	
	/**********Initiating Phase 1*********/
//	cutilCheckError(cutStartTimer(timer));
	sdkStartTimer(&timer);
	//launch phase 1 , bucket allocation
	phase12<<<length/512+1,512>>>(d_keys,d_offset,length,d_count,*bucketCount);
	CheckCUDAError();
//	cutilCheckError(cutStopTimer(timer));
	sdkStopTimer(&timer);

	/************  Calculating Start of each bucket (prefix sum of Count) **********/
	//allocate and initiazlie start 
	checkCudaErrors( hipMalloc( (void**) &d_start, bucketDataSize));
	checkCudaErrors( hipMemset(d_start,0,bucketDataSize));

	//find prefix sum 
	CUDPPConfiguration config;
    config.op = CUDPP_ADD;
	config.datatype = CUDPP_UINT;
    config.algorithm = CUDPP_SCAN;
    config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;

    CUDPPHandle scanplan = 0;
    CUDPPResult result = cudppPlan(&scanplan, config, *bucketCount, 1, 0);
//	cutilCheckError(cutStartTimer(timer));
	sdkStartTimer(&timer);
	// Run the scan
    cudppScan(scanplan, d_start, d_count, *bucketCount);
    CheckCUDAError();
//	cutilCheckError(cutStopTimer(timer));
	sdkStopTimer(&timer);
	cudppDestroyPlan(scanplan);
	

	/************* Copying to buffer **************/

	
	//allocate buffer
	checkCudaErrors( hipMalloc( (void**) &d_bufferK, length*KEY_SIZE));
	checkCudaErrors( hipMalloc( (void**) &d_bufferV, length*VALUE_SIZE));
//	cutilCheckError(cutStartTimer(timer));
	sdkStartTimer(&timer);
	//copy to buckets
	copyToBucket2<<<length/512+1,512>>>(d_keys,d_values,d_offset,length,d_start,*bucketCount,d_bufferK,d_bufferV);
	CheckCUDAError();
//	cutilCheckError(cutStopTimer(timer));
	sdkStopTimer(&timer);
	
	//free up some resources
	checkCudaErrors(hipFree(d_offset));

	/***************     Cuckoo Hashing        ******************/
	checkCudaErrors( hipMalloc( (void**) d_bucketSeed, bucketDataSize));
	checkCudaErrors( hipMalloc( (void**) d_TK, (*bucketCount)*BUCKET_KEY_SIZE));
	checkCudaErrors( hipMalloc( (void**) d_TV, (*bucketCount)*BUCKET_VALUE_SIZE));

	unsigned int randomSeedSize=*bucketCount*MAX_SEED_COUNT* sizeof(int);
	h_randomSeed=( unsigned int *)  malloc(randomSeedSize);
	checkCudaErrors( hipMalloc( (void**) &d_randomSeed,randomSeedSize));
	for(unsigned int i=0;i<*bucketCount;i++){
		for(int j=0;j<MAX_SEED_COUNT;j++){
			*(h_randomSeed+i*MAX_SEED_COUNT+j)=rand();
		}
	}
	checkCudaErrors( hipMemcpy( d_randomSeed, h_randomSeed, randomSeedSize,hipMemcpyHostToDevice) );
	free(h_randomSeed);


	//phase2<<<*bucketCount,512>>>(d_buffer,d_start,d_count,*d_bucketSeed,d_bucketState,*bucketCount,*d_T,d_randomSeed,MAX_SEED_COUNT);
	//phase22<<<*bucketCount,512>>>(d_bufferK,d_bufferV,d_start,d_count,*d_bucketSeed,*bucketCount,*d_TK,*d_TV,d_randomSeed,MAX_SEED_COUNT);
	phase2HostLaunch(d_bufferK,d_bufferV,
			d_start, d_count,
			*d_bucketSeed,*bucketCount,
			*d_TK,*d_TV,
			MAX_SEED_COUNT,length);
	CheckCUDAError();
//	cutilCheckError(cutStopTimer(timer));
	sdkStopTimer(&timer);
	hipError_t err=hipGetLastError();
	if(hipSuccess != err ){
		printf("%s\n",hipGetErrorString(err));
	}
	checkCudaErrors(hipFree(d_start));
		checkCudaErrors(hipFree(d_count));

	checkCudaErrors(hipFree(d_randomSeed));
	checkCudaErrors(hipFree(d_bufferK));
	checkCudaErrors(hipFree(d_bufferV));

	
	
	*tableLength=*bucketCount*BLOCK_SIZE;

	DEBUG_CALL(verifyHashTable2(d_keys,d_values,length,*d_TK,*d_TV,*tableLength,*d_bucketSeed,*bucketCount));
//	cutilCheckError(cutDeleteTimer(timer));
	sdkDeleteTimer(&timer);
}
