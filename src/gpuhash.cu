#include "hip/hip_runtime.h"
#define CUDPP_STATIC_LIB



#include "cutil_inline.h"
#include "cudpp.h"
#include "utils.h"
#include "gpuhash_device.h"
#include "gpuhash_kernel.cu"
#include <time.h>


#ifdef EULER_NDEBUG
#define DEBUG_GPUHASH_CU(x)
#else
#define DEBUG_GPUHASH_CU(x) x
#endif
#define DEBUG_CALL(x) DEBUG_GPUHASH_CU(x)


void bucketSortHost(KEY_PTR   bufferK,VALUE_PTR bufferV, unsigned int * start,unsigned int * bucketSize,unsigned int bucketCount,KEY_PTR TK,VALUE_PTR TV,unsigned int bid,unsigned int tid){

		KEY_T keys[512];
		unsigned int keyCount=0;
		unsigned int blockOffset=start[bid];
		unsigned int size=bucketSize[bid];
		if (tid< size) 
			keys[tid]=bufferK[blockOffset+tid];// 
//		__syncthreads();
		if(tid<size){
			for(int i=0; i<size; i++){
				if( keys[tid]> keys[i] ) keyCount++;
			}
		}
		//__syncthreads();
		
		if(tid<size) {
			TK[GET_KEY_INDEX(bid,keyCount)]=keys[tid];
			TV[GET_VALUE_INDEX(bid,keyCount)]=bufferV[blockOffset+tid];
		}
}


void verifyBuffer(KEY_PTR d_keys,VALUE_PTR d_values,unsigned int length, KEY_PTR d_bufferK, VALUE_PTR d_bufferV, unsigned int * d_bucketOffset,unsigned int * d_bucketSize, unsigned int bucketCount){

	KEY_PTR h_keys;
	VALUE_PTR h_values;
	KEY_PTR	h_bufferK;
	VALUE_PTR h_bufferV;
	unsigned int * h_bucketOffset;
	unsigned int * h_bucketSize;
	
	
	h_keys=(KEY_PTR) malloc(length * KEY_SIZE);
	h_values=(VALUE_PTR) malloc( length * VALUE_SIZE);
	h_bufferK= (KEY_PTR) malloc ( length *KEY_SIZE);
	h_bufferV= (VALUE_PTR) malloc( length * VALUE_SIZE);
	h_bucketOffset =(unsigned int *) malloc( bucketCount * sizeof(unsigned int));
	h_bucketSize = (unsigned int * ) malloc ( bucketCount * sizeof(unsigned int));	
	
	cutilSafeCall( hipMemcpy(h_keys, d_keys, length * (KEY_SIZE), hipMemcpyDeviceToHost));
	cutilSafeCall( hipMemcpy(h_values, d_values, length * (VALUE_SIZE), hipMemcpyDeviceToHost));
	cutilSafeCall( hipMemcpy(h_bufferK, d_bufferK, length * (KEY_SIZE), hipMemcpyDeviceToHost));
	cutilSafeCall( hipMemcpy(h_bufferV, d_bufferV, length * (VALUE_SIZE), hipMemcpyDeviceToHost));
	cutilSafeCall( hipMemcpy(h_bucketOffset, d_bucketOffset, bucketCount* (sizeof(unsigned int)), hipMemcpyDeviceToHost));
	cutilSafeCall( hipMemcpy(h_bucketSize, d_bucketSize, bucketCount * (sizeof(unsigned int)), hipMemcpyDeviceToHost));

	unsigned int found=0;
	unsigned int notfound=0;
	unsigned int correctValue=0;
	unsigned int incorrectValue=0;

	for (unsigned int i=0; i<length ; i++){
		unsigned int j=0;
		unsigned int bucket= host_hash_h(h_keys[i],bucketCount);
		unsigned int offset=h_bucketOffset[bucket];
		unsigned int size = h_bucketSize[bucket];
	
		while( j<size && h_bufferK[offset+j]!=h_keys[i]) j++;
		if( j<size) {
			found ++;
			if( h_values[i]== h_bufferV[offset+j]) {
				correctValue++;
			}else {
				incorrectValue++;
			}
		}
		else {
			notfound++;
			incorrectValue++;
		}
		
	}
	printf("found:[%u], notfound:[%u] , correct:[%u] ,incorrect:[%u]\n",found,notfound,correctValue,incorrectValue);
	free(h_keys);
	free(h_values);
	free(h_bufferK);
	free(h_bufferV);
	free(h_bucketOffset);
	free(h_bucketSize);
}/*
unsigned int  host_hash_h(KEY_T key, unsigned int bucketCount){
	return ((C0+C1*key)% LARGE_PRIME )% bucketCount;
}*/
unsigned int host_hash_g1(KEY_T key,unsigned int seed){
	return ((C10^seed+(C11^seed)*key)% LARGE_PRIME )%L2_SIZE;
}
unsigned int host_hash_g2(KEY_T key,unsigned int seed){
	return ((C20^seed+(C21^seed)*key)% LARGE_PRIME )%L2_SIZE;
}
unsigned int host_hash_g3(KEY_T  key,unsigned int seed){
	return ((C30^seed+(C31^seed)*key)% LARGE_PRIME )%L2_SIZE;
}
/*

VALUE_T host_getHashValue(KEY_T key, KEY_PTR TK,VALUE_PTR TV,unsigned int * bucketSize, unsigned int bucketCount,unsigned int * bucket){
	*bucket=host_hash_h(key,bucketCount);

	unsigned int l=0;
	unsigned int r=bucketSize[*bucket];
	unsigned int mid;
	while(l<r){
		mid =l+((r-l)/2);
		//if( GET_HASH_KEY(T,(*bucket),mid) <key) {
		if( TK[GET_KEY_INDEX(*bucket,mid)] <key) {
			l=mid+1;
		}else {
			r=mid;
		}
	}
	//if(l < bucketSize[*bucket] && (GET_HASH_KEY(T,(*bucket),l))==key){ 
	if(l < bucketSize[*bucket] && TK[GET_KEY_INDEX(*bucket,l)]==key){ 
		//return GET_HASH_VALUE(T,(*bucket),l);
		return TV[GET_VALUE_INDEX(*bucket,l)];
	}else {
		printf("value not found\nprinting bucket data");
		for(unsigned int i =0;i<bucketSize[*bucket];i++){
			printf("[%u]:{%lu}=>%u\t\t",i,(unsigned long )(TK[GET_KEY_INDEX((*bucket),i)]),(unsigned int )(TV[GET_VALUE_INDEX((*bucket),i)]));
		}
		printf("\n");
		return MAX_INT;
	}
}
*/
void verifyHashTable(KEY_PTR  d_keys, VALUE_PTR d_values, unsigned int length, KEY_PTR  d_TK,VALUE_PTR d_TV ,unsigned int tableLength, unsigned int * d_bucketSize, unsigned int bucketCount){

	KEY_PTR		 h_keys;
	VALUE_PTR	 h_values;
	KEY_PTR		 h_TK;
	VALUE_PTR	 h_TV;
	unsigned int * 	 h_bucketSize;
	
	unsigned int *	 bCount;
	unsigned int b;
	
	h_keys=(KEY_PTR) malloc(length * (KEY_SIZE));
	h_values=(VALUE_PTR) malloc(length * (VALUE_SIZE));
	h_TK= (KEY_PTR ) malloc( BUCKET_KEY_SIZE* bucketCount);
	h_TV= (VALUE_PTR ) malloc( BUCKET_VALUE_SIZE* bucketCount);
	h_bucketSize= (unsigned int *) malloc ( bucketCount * sizeof(unsigned int));
	bCount= (unsigned int * ) malloc( bucketCount *sizeof(unsigned int));
	
	cutilSafeCall( hipMemcpy(h_keys, d_keys, length * (KEY_SIZE), hipMemcpyDeviceToHost));
	cutilSafeCall( hipMemcpy(h_values, d_values, length * (VALUE_SIZE), hipMemcpyDeviceToHost) );
	cutilSafeCall( hipMemcpy(h_TK, d_TK,  BUCKET_KEY_SIZE*bucketCount, hipMemcpyDeviceToHost) );
	cutilSafeCall( hipMemcpy(h_TV, d_TV,  BUCKET_VALUE_SIZE*bucketCount, hipMemcpyDeviceToHost) );
	cutilSafeCall( hipMemcpy(h_bucketSize, d_bucketSize, bucketCount * sizeof(unsigned int), hipMemcpyDeviceToHost) );
	
	CheckCUDAError();
	/*for(int j=0; j<bucketCount; j++){
		printf(" [%u]:%u ,",h_bucketSeed[j],j);
		}
		printf("\n");*/
	for(int j=0; j<bucketCount; j++){
		if(h_bucketSize[j] >=500) printf("possible invalid bucket [%u] size[%u]\n",j,h_bucketSize[j]);
		/*printf("BUCKET -> [%u]\n",j);
		for (int l=0; l<3; l++){
			printf("TABLE : T%d\n",l);
			for (int k=0; k< L2_SIZE; k++){
				printf("[%u]={%u}, ",h_T[j*BLOCK_SIZE + L2_SIZE*2*l +k*2],h_T[j*BLOCK_SIZE + L2_SIZE*2*l +k*2+1]);
			}
			printf("\n");
		}*/
	}
	unsigned int correct=0;
	unsigned int incorrect=0;
	
	memset(bCount,0,bucketCount * sizeof(unsigned int) );
	for (int i=0;i <length; i++){
		if( host_getHashValue(h_keys[i],h_TK,h_TV,h_bucketSize,bucketCount,&b) == h_values[i]) { 
			correct++;
			bCount[b]++;
		}
		else incorrect++;
	}
	printf("total verified %u, incorrect %u\n",correct,incorrect);
	
	//for(int  i =0; i<bucketCount; i++){ printf(" Count Bucket-%d=%u\n",i,bCount[i]);}
	free(h_keys);
	free(h_values);
	free(h_TK);
	free(h_TV);
	free(h_bucketSize);
	free(bCount);
	
}

/**	Create Hash table with linear probing sorting the buckets and performing Binary Search on Lookup****/
extern "C"
void createHashTable(KEY_PTR d_keys,VALUE_PTR d_values, unsigned int length, KEY_PTR *  d_TK,VALUE_PTR * d_TV,unsigned int * tableLength, unsigned int ** d_bucketSize,unsigned int * bucketCount){
		
	unsigned int * d_offset;	
	unsigned int * d_start;
	
	KEY_PTR d_bufferK;
	VALUE_PTR d_bufferV;
	

	unsigned int timer = 0;
	cutilCheckError(cutCreateTimer(&timer));
	srand ( time(NULL) );
	*bucketCount=(length /409)+1; //ceil
	unsigned int dataSize=length*sizeof(unsigned int);
	unsigned int bucketDataSize=*bucketCount*sizeof(unsigned int);

	cutilSafeCall( hipMalloc( (void**) &d_offset, dataSize));
	//allocate count 
	cutilSafeCall( hipMalloc( (void**) d_bucketSize, bucketDataSize));
	
	
	//initialize offset to zero
	cutilSafeCall( hipMemset(d_offset,0,dataSize));
	//initialize count to zero
	cutilSafeCall( hipMemset(*d_bucketSize,0,bucketDataSize));
	
	
	/**********Initiating Phase 1*********/
	cutilCheckError(cutStartTimer(timer));
	//launch phase 1 , bucket allocation
	phase1<<<length/512+1,512>>>(d_keys,d_offset,length,*d_bucketSize,*bucketCount);
	CheckCUDAError();
	cutilCheckError(cutStopTimer(timer));
	/************  Calculating Start of each bucket (prefix sum of Count) **********/
	//allocate and initiazlie start 
	cutilSafeCall( hipMalloc( (void**) &d_start, bucketDataSize));
	cutilSafeCall( hipMemset(d_start,0,bucketDataSize));

	//find prefix sum 
	CUDPPConfiguration config;
    	config.op = CUDPP_ADD;
	config.datatype = CUDPP_UINT;
    	config.algorithm = CUDPP_SCAN;
    	config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;
  
	CUDPPHandle scanplan = 0;
    	CUDPPResult result = cudppPlan(&scanplan, config, *bucketCount, 1, 0);
	cutilCheckError(cutStartTimer(timer));
	// Run the scan
    	cudppScan(scanplan, d_start, *d_bucketSize, *bucketCount);
    	CheckCUDAError();
	cutilCheckError(cutStopTimer(timer));
	cudppDestroyPlan(scanplan);
	

	/************* Copying to buffer **************/

	
	//allocate buffer
	cutilSafeCall( hipMalloc( (void**) &d_bufferK, length*KEY_SIZE));
	cutilSafeCall( hipMalloc( (void**) &d_bufferV, length*VALUE_SIZE));
	cutilCheckError(cutStartTimer(timer));
	//copy to buckets
	copyToBucket<<<length/512+1,512>>>(d_keys,d_values,d_offset,length,d_start,*bucketCount,d_bufferK,d_bufferV);
	CheckCUDAError();
	cutilCheckError(cutStopTimer(timer));
	
	//verifyBuffer(d_keys,d_values,length,d_bufferK,d_bufferV,d_start,*d_bucketSize,*bucketCount);

	
	//free up some resources
	cutilSafeCall(hipFree(d_offset));
	
	/***************     Cuckoo Hashing        ******************/
	
	cutilSafeCall( hipMalloc( (void**) d_TK, (*bucketCount)*BUCKET_KEY_SIZE));
	cutilSafeCall( hipMalloc( (void**) d_TV, (*bucketCount)*BUCKET_VALUE_SIZE));
	bucketSort<<<*bucketCount,32>>>(d_bufferK,d_bufferV,d_start,*d_bucketSize,*bucketCount,*d_TK,*d_TV);
	CheckCUDAError();
	cutilCheckError(cutStopTimer(timer));
	hipError_t err=hipGetLastError();
	if(hipSuccess != err ){
		printf("%s\n",hipGetErrorString(err));
	}
	
//	verifyBuffer(d_keys,d_values,length,d_bufferK,d_bufferV,d_start,*d_bucketSize,*bucketCount);
	cutilSafeCall(hipFree(d_start));
	cutilSafeCall(hipFree(d_bufferK));
	cutilSafeCall(hipFree(d_bufferV));
	
	*tableLength=*bucketCount*MAX_BUCKET_ITEM ;

	DEBUG_CALL(verifyHashTable(d_keys,d_values,length,*d_TK,*d_TV,*tableLength,*d_bucketSize,*bucketCount));
	cutilCheckError(cutDeleteTimer(timer));
	
}
