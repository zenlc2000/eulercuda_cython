#include "hip/hip_runtime.h"


#include "common.h"



__global__ void phase1(	KEY_PTR  keys,
			unsigned int * offset,
			unsigned int length,
			unsigned int* count,
			unsigned int bucketCount)
{

	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	if(tid<length)
	{
		KEY_T key=keys[tid];
		unsigned int bucket=hash_h(key,bucketCount);
		offset[tid]=atomicInc (count+bucket,MAX_INT);

	}
	__syncthreads();
}


__global__ void copyToBucket(	KEY_PTR keys,
				VALUE_PTR values,
				unsigned int * offset,
				unsigned int length,
				unsigned int* start,
				unsigned int bucketCount,
				KEY_PTR  bufferK,
				VALUE_PTR bufferV)
{

	unsigned tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;

	if(tid<length)
	{
		KEY_T key =keys[tid];
		unsigned int bucket=hash_h(key,bucketCount);
		VALUE_T value=values[tid];
		unsigned int index=start[bucket]+offset[tid];
		//index=(index * BUCKET_ITEM_SIZE);
		bufferK[index]=key;
		bufferV[index]=value;
		//*(BUFFER_ITEM_KEY_PTR(buffer,index))=key;
		//*(BUFFER_ITEM_VALUE_PTR(buffer,index))=value;
	}
}
__global__ void bucketSort(KEY_PTR   bufferK,VALUE_PTR bufferV, unsigned int * start,unsigned int * bucketSize,unsigned int bucketCount,KEY_PTR TK,VALUE_PTR TV){


		__shared__ KEY_T keys[MAX_BUCKET_ITEM];
		unsigned int keyCount[MAX_BUCKET_ITEM/32];
		//unsigned int keyCount=0;
		unsigned int blockOffset=start[blockIdx.x];
		unsigned int size=bucketSize[blockIdx.x];

		unsigned int chunks=size>>5;
		chunks= (chunks<<5==size)?chunks:chunks+1;
		for(unsigned int j=0;j<chunks;j++){
			if((j<<5)+threadIdx.x<size)
				keys[(j<<5)+threadIdx.x]=bufferK[blockOffset+(j<<5)+threadIdx.x];//
		}

		__syncthreads();
		for(unsigned int j=0;j<chunks;j++){
			if((j<<5)+threadIdx.x<size){
				keyCount[j]=0;
				for(int i=0; i<size; i++){
					//if( keys[(i<<5)+threadIdx.x]> keys[i] ) keyCount++;
					keyCount[j]=( keys[(j<<5)+threadIdx.x]> keys[i] )?keyCount[j]+1:keyCount[j];
				}
			}
		}
			__syncthreads();
		for(unsigned int j=0;j<chunks;j++){
			if((j<<5)+threadIdx.x<size){
				TK[GET_KEY_INDEX(blockIdx.x,keyCount[j])]=keys[(j<<5)+threadIdx.x];
				TV[GET_VALUE_INDEX(blockIdx.x,keyCount[j])]=bufferV[blockOffset+(j<<5)+threadIdx.x];
			}
		}
}
