#include "hip/hip_runtime.h"
#define CUDPP_STATIC_LIB
#include <algorithm>
#include "graph.h"
//#include <cutil_inline.h>
#include "/Volumes/Macintosh HD/Developer/NVIDIA/CUDA-7.5/samples/common/inc/hip/hip_runtime_api.h" 
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "utils.h"
#include "cudpp.h"
#include "component.h"
#ifdef EULER_NDEBUG
#define DEBUG_EULER_CU(x)
#else
#define DEBUG_EULER_CU(x) x
#endif
//#define DEBUG_EULER_CU(x) x
#define DEBUG_CALL(x)  DEBUG_EULER_CU(x)

void printSuccessorGraph(Vertex * d_v , unsigned int length){
	
	Vertex * h_v =NULL;
	h_v=(Vertex * ) malloc(length* sizeof(Vertex));
	checkCudaErrors(hipMemcpy(h_v,d_v,length*sizeof(Vertex),hipMemcpyDeviceToHost));
	printf("$graph G {\n");
	for (unsigned int i =0; i< length; i++){	
		if(h_v[i].n1 < length)	printf("$\t%u -- %u\n",h_v[i].vid, h_v[i].n1);
		if(h_v[i].n2 < length)  printf("$\t%u -- %u\n",h_v[i].vid, h_v[i].n2);
	}
	printf("$}\n");
	free(h_v);
}
void printCircuitGraph(CircuitEdge * d_ce , unsigned int length){
	
	CircuitEdge * h_ce =NULL;
	h_ce=(CircuitEdge * ) malloc(length* sizeof(CircuitEdge));
	checkCudaErrors(hipMemcpy(h_ce,d_ce,length*sizeof(CircuitEdge),hipMemcpyDeviceToHost));
	printf("$graph G {\n");
	for (unsigned int i =0; i< length; i++){	
		printf("$\t%u -- %u [ label= e1:%u-e2:%u ]\n",h_ce[i].c1,h_ce[i].c2,h_ce[i].e1,h_ce[i].e2);
	}
	printf("$}\n");
	free(h_ce);
}
/*** Assig Successor**/

__global__  void assignSuccessor(EulerVertex * ev,unsigned int * l, unsigned int * e, unsigned vcount, EulerEdge * ee ,unsigned int ecount){
	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	unsigned int eidx=0;
	if(tid<vcount){		
		while(eidx<ev[tid].ecount && eidx<ev[tid].lcount){
			ee[e[ev[tid].ep+eidx]].s=l[ev[tid].lp+eidx] ;
			eidx++;
		}
	}
}
void validateSuccessors(EulerEdge * d_ee, unsigned int ecount) {
	EulerEdge * h_ee;

	h_ee= (EulerEdge * ) malloc( sizeof(EulerEdge) *ecount);
	checkCudaErrors(hipMemcpy(h_ee,d_ee,ecount * sizeof(EulerEdge),hipMemcpyDeviceToHost));
	
	unsigned int snot=0;
	for(unsigned int i =0;i< ecount; i++){
		if( h_ee[i].s==ecount) {snot++;}
	}
	printf("total edges with succesors not set :%u\n",snot);
	free(h_ee);
}

/** Constuct Succesor Graph**/ //Redundant
__global__ void constructSuccessorGraphP1(EulerEdge* e, Vertex * v, unsigned int ecount){
	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	
	if(tid<ecount){
		v[tid].n1=ecount;v[tid].n2=ecount;//v[tid].n3=ecount;v[tid].n4=ecount;
		v[tid].vid=e[tid].eid;
		v[tid].n1=e[tid].s;
	}
}

__global__ void constructSuccessorGraphP2(EulerEdge* e, Vertex * v, unsigned int ecount){
	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	
	if(tid<ecount){
	/*	if(v[v[tid].n1].n1 < ecount){
			v[v[tid].n1].n2=v[tid].vid;
		}else{
			v[v[tid].n1].n1=v[tid].vid;
		}*/
		if(v[tid].n1 <ecount ){
			v[v[tid].n1].n2=v[tid].vid;
		}
	}
}
/***   Calculate Circuit Graph Vertex  ***/
__global__ void calculateCircuitGraphVertexData( unsigned int * D,unsigned int * C,unsigned int ecount){
	
	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	if( tid <ecount)
	{
		unsigned int c=D[tid];
		atomicExch(C+c,1);
	}	
}
/*** construct circuit graph vertex **/
__global__ void constructCircuitGraphVertex(unsigned int * C,unsigned int * offset,unsigned int ecount, unsigned int * cv, unsigned int cvCount){
	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	if(tid < ecount){
		if(C[tid]!=0){
			cv[offset[tid]]=tid;
		}
	}
}

/*** Calculate Circuit Graph Edges***/
__global__ void calculateCircuitGraphEdgeData(EulerVertex* v,unsigned int * e,unsigned vCount,unsigned int * D,unsigned int * map,unsigned int ecount, unsigned int * cedgeCount/*, unsigned int cvCount*/){
	
	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	unsigned int index=0;
	unsigned int maxIndex=0;
	index=0;
	maxIndex=0;
	if(tid<vCount && v[tid].ecount>0 ){
		index=v[tid].ep;
		maxIndex=index+v[tid].ecount-1;
		while (index < maxIndex ){
			unsigned int c1=map[D[e[index]]];
			unsigned int c2=map[D[e[index+1]]];
			if( c1 !=c2){
				unsigned int c=min(c1,c2);
				atomicInc(cedgeCount+c,ecount);
			}
	
			index++;
		}
	}

}
__global__ void assignCircuitGraphEdgeData(EulerVertex* v,
					   unsigned int * e,
					   unsigned vCount,
					   unsigned int * D,
					   unsigned int * map,
					   unsigned int ecount, 
					   unsigned int * cedgeOffset,
					   unsigned int * cedgeCount, 
					   unsigned int cvCount, 
					   CircuitEdge * cedge,  
					   unsigned int cecount){

	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	unsigned int index=0;
	unsigned int maxIndex=0;
	if(tid<vCount && v[tid].ecount>0){
		index=v[tid].ep;
		maxIndex=index+v[tid].ecount-1;
		while (index<maxIndex   ){			
			unsigned int c1=map[D[e[index]]];
			unsigned int c2=map[D[e[index+1]]];
			if( c1 !=c2){
				unsigned int c=min(c1,c2);
				unsigned int t=max(c1,c2);
				unsigned int i=atomicDec(cedgeCount+c,ecount);
				i=i-1;
				cedge[cedgeOffset[c]+i].c1=c;
				cedge[cedgeOffset[c]+i].c2=t;
				cedge[cedgeOffset[c]+i].e1=e[index];
				cedge[cedgeOffset[c]+i].e2=e[index+1];
			}				
			index++;
		}
	}
}

/*
__global__ void markSegments(	unsigned short * d_mark,
				unsigned int 	circuitGraphEdgeCount,
				unsigned int * 	d_cg_edge_start,
				unsigned int *	d_cedgeCount,
				unsigned int 	circuitVertexSize){

	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	if(tid<circuitVertexSize){
		d_mark[ d_cg_edge_start[tid]]=d_cedgeCount[tid];
	}
	
}
*/
/*
__global__ void sortCircuit(	unsigned int cedgeOffset,
				unsigned int cedgeCount,
				unsigned int circuitCount,
				CircuitEdge * cedge ){

	unsigned int bid=0;
	unsigned int tid=0;
	
	unsigned int keyCount=0;
	unsigned int offset=cedgeOffset[bid];
	unsigned int itemCount=circuitCount[bid];
	unsigned int chunks=itemCount/256;
	unsigned int chunkSize=0;
	unsigned int chunkIdx=0;
	__shared__ unsigned int keys[256];

	if(bid<circuitCount){
		while(chunkIdx<Chunks){
			if(tid<	itemCount)
				keys[tid]=edge[offset+tid].e2;
			__syncthreads();
			if(tid<itemCount){
				for(int i=0;i<256;i++){
					if(keys[tid]>keys[i]) keyCount++;
				}
			}
			__syncthreads();
			CircuitEdge temp;
			if(tid<itemCount){
				temp=cedge[tid];			
			}
			__syncthreads();
		
		}
	
	}
}*/


__device__ unsigned int getValue(CircuitEdge cedge, unsigned char radix){

	switch(radix){
		case 0: return cedge.e2;
		case 1: return cedge.e1;
		case 2: return cedge.c2;
	}
	return 0xFFFFFFFF;
}

/*
__global__ void sortCircuitGraphEdgeData3( unsigned int * cedgeOffset,
					   unsigned int * cedgeCount, 
					   unsigned int circuitCount, 
					   CircuitEdge * cedge,
					   unsigned short * mark,
					   unsigned int edgeCount,
					   unsigned char radix){


	unsigned int chunks=blockDim.x;
	unsigned int chunkSize=cedgeCount[bid]/chunks; //fix off by 1
	unsigned int offset=cedgeOffset[bid]+chunkSize*threadIdx.x;

	//now scan
	while(mark[offset]==0 && offset<cedgeCount[bid]) offset++;

	//__syncthreads();

	//everyone looking at its own chunk and we have to sort (mark[Offset] sized data)
	unsigned int count=mark[Offset];
	for(int i=0;i<count;i++){
		minIndex=offset+i;
		minValue=getValue(cedge[minIndex],radix);// cedge[minIndex].c2;
		for( j=offset+i+1;j<offset+count;j++){
			unsigned int nextValue=getValue(cedge[j],radix);
			if( minValue > nextValue){
				minIndex=j;
				minValue=nextValue;
			}
		}
		if(minIndex != offset+i){
			CircuitEdge temp=cedge[offset+i];
			cedge[offset+i]=cedge[minIndex];
			cedge[minIndex]=temp;
		}
	}
	mark[Offset]=0;
	offset+=count;
	//scan onemore time to count same
	
	


	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	unsigned int minIndex=0;
	unsigned int minValue=0;
	unsigned int i =0;
	unsigned int j=0;
	unsigned int count;
	unsigned int offset=0;

	if(tid<circuitCount){
		count=cedgeCount[tid];
		offset=cedgeOffset[tid];
		for (i=0;i<count;i++){
			minIndex=offset+i;
			minValue=getValue(cedge[minIndex],radix);// cedge[minIndex].c2;
			for( j=offset+i+1;j<offset+count;j++){
				unsigned int nextValue=getValue(cedge[j],radix);
				if( minValue > nextValue){
					minIndex=j;
					minValue=nextValue;
				}
			}
			if(minIndex != offset+i){
				CircuitEdge temp=cedge[offset+i];
				cedge[offset+i]=cedge[minIndex];
				cedge[minIndex]=temp;
			}
		}
	}
	
	
}
*/

__global__ void sortCircuitGraphEdgeData2( unsigned int * cedgeOffset,
					   unsigned int * cedgeCount, 
					   unsigned int circuitCount, 
					   CircuitEdge * cedge,unsigned char radix){

	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	unsigned int minIndex=0;
	unsigned int minValue=0;
	unsigned int i =0;
	unsigned int j=0;
	unsigned int count;
	unsigned int offset=0;

	if(tid<circuitCount){
		count=cedgeCount[tid];
		offset=cedgeOffset[tid];
		for (i=0;i<count;i++){
			minIndex=offset+i;
			minValue=getValue(cedge[minIndex],radix);// cedge[minIndex].c2;
			for( j=offset+i+1;j<offset+count;j++){
				unsigned int nextValue=getValue(cedge[j],radix);
				if( minValue > nextValue){
					minIndex=j;
					minValue=nextValue;
				}
			}
			if(minIndex != offset+i){
				CircuitEdge temp=cedge[offset+i];
				cedge[offset+i]=cedge[minIndex];
				cedge[minIndex]=temp;
			}
		}
	}/*
		}
	*/
}

__global__ void sortCircuitGraphEdgeData( unsigned int * cedgeOffset,
					   unsigned int * cedgeCount, 
					   unsigned int circuitCount, 
					   CircuitEdge * cedge){

	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;
	unsigned int minIndex=0;
	unsigned int minValue=0;
	unsigned int i =0;
	unsigned int j=0;
	unsigned int count;
	unsigned int offset=0;

	if(tid<circuitCount){
		count=cedgeCount[tid];
		offset=cedgeOffset[tid];
		for (i=0;i<count;i++){
			minIndex=offset+i;
			minValue=cedge[minIndex].c2;
			for( j=offset+i+1;j<offset+count;j++){
				if(minValue > cedge[j].c2){
					minIndex=j;
					minValue=cedge[j].c2;
				}else if( minValue == cedge[j].c2){
					if( cedge[minIndex].e1> cedge[j].e1){
						minIndex=j;
						minValue=cedge[j].c2;
					}else if(cedge[minIndex].e1 == cedge[j].e1){
						if(cedge[minIndex].e2 > cedge[j].e2) {
							minIndex=j;
							minValue=cedge[j].c2;
						}
					}
				}
			}
			if(minIndex != offset+i){
				CircuitEdge temp=cedge[offset+i];
				cedge[offset+i]=cedge[minIndex];
				cedge[minIndex]=temp;
			}
		}
	}/*
	if(tid<vCount && v[tid].ecount>0){
		index=v[tid].ep;
		maxIndex=index+v[tid].ecount-1;
		while (index<maxIndex   ){			
			unsigned int c1=map[D[e[index]]];
			unsigned int c2=map[D[e[index+1]]];
			if( c1 !=c2){
				unsigned int c=min(c1,c2);
				unsigned int t=max(c1,c2);
				unsigned int i=atomicDec(cedgeCount+c,ecount);
				i=i-1;
				cedge[cedgeOffset[c]+i].c1=c;
				cedge[cedgeOffset[c]+i].c2=t;
				cedge[cedgeOffset[c]+i].e1=e[index];
				cedge[cedgeOffset[c]+i].e2=e[index+1];
			}				
			index++;
		}
	}*/
}
__global__  void identifyContigStart( EulerEdge * e ,unsigned char * contigStart,unsigned int ecount){
	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;	
	if(tid<ecount){
		if(e[tid].s < ecount){
			contigStart[e[tid].s]=0;
			//atomicExch(contigStart+e[tid].s,0);
		}
	}
}

__global__ void  markSpanningEulerEdges(EulerEdge * ee, unsigned int * mark , unsigned int ecount,CircuitEdge * cg_edge,unsigned int cg_edgeCount,unsigned int * tree, unsigned int treeCount){

	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;	
	if(tid < treeCount) {
		/*if(tree[tid]==1)*/{
			atomicExch(mark+min(cg_edge[tree[tid]].e1,cg_edge[tree[tid]].e2),1); // important: assumption if(mark[i]=1) means mark[i]and mark[i+1] are swipe
			//atomicExch(mark+cg_edge[tree[tid]].e2,1);
			
		}
	}
}

__global__ void executeSwipe(EulerVertex * ev,unsigned int * e, unsigned int vcount , EulerEdge * ee, unsigned int * mark,unsigned int ecount){

	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;	
	unsigned int t;
	unsigned int index=0;
	unsigned int maxIndex;
	unsigned int s;
	if( tid< vcount){
		index=ev[tid].ep;
		maxIndex=index+ev[tid].ecount-1;
		while( index<maxIndex){

			if(mark[ee[e[index]].eid]==1){
				t=index;
				s=ee[e[index]].s;
				while(mark[ee[e[index]].eid]==1 && index < maxIndex){					
					ee[e[index]].s=ee[e[index+1]].s;
					index=index+1;
				}
				if(t!=index){
					ee[e[index]].s=s;
				}
			}
			index++;
		}

	}
}

 void executeSwipeHost(EulerVertex * ev,unsigned int * e, unsigned int vcount , EulerEdge * ee, unsigned int * mark,unsigned int ecount, unsigned int tid){

//	unsigned int tid=(blockDim.x*blockDim.y * gridDim.x*blockIdx.y) + (blockDim.x*blockDim.y*blockIdx.x)+(blockDim.x*threadIdx.y)+threadIdx.x;	
	unsigned int t;
	unsigned int index=0;
	unsigned int maxIndex;
	unsigned int s;
	if( tid< vcount){
		index=ev[tid].ep;
		maxIndex=index+ev[tid].ecount-1;
		while( index<maxIndex){

			if(mark[ee[e[index]].eid]==1){
				t=index;
				s=ee[e[index]].s;
				while(mark[ee[e[index]].eid]==1 && index < maxIndex){					
					ee[e[index]].s=ee[e[index+1]].s;
					index=index+1;
				}
				if(t!=index){
					ee[e[index]].s=s;
				}
			}
			index++;
		}

	}
}
void executeSwipeHostLaunch(EulerVertex * d_ev, unsigned int * d_e, unsigned int vcount, EulerEdge * d_ee, unsigned int * d_mark , unsigned int ecount){

		EulerVertex *  h_ev;
		unsigned int * h_e;
		EulerEdge * h_ee;
		unsigned int * h_mark;
		
		logMessage(LOG_LVL_DETAIL,"executeSwipeHostLaunch");
		h_ev=(EulerVertex *)malloc(vcount*sizeof(EulerVertex));	
		h_e=(unsigned int *) malloc(vcount * sizeof(unsigned int ));
		h_ee =(EulerEdge *) malloc(ecount * sizeof(EulerEdge));
		h_mark=(unsigned int *) malloc(ecount * sizeof(EulerEdge));
		
		checkCudaErrors( hipMemcpy(h_ev,d_ev, vcount*sizeof(EulerVertex), hipMemcpyDeviceToHost));
		checkCudaErrors( hipMemcpy(h_e,d_e, vcount*sizeof(unsigned int), hipMemcpyDeviceToHost));
		checkCudaErrors( hipMemcpy(h_ee,d_ee, ecount*sizeof(EulerEdge), hipMemcpyDeviceToHost));
		checkCudaErrors( hipMemcpy(h_mark,d_mark, ecount*sizeof(unsigned int), hipMemcpyDeviceToHost));
		
		for(unsigned tid =0;tid<vcount;tid++){
			executeSwipeHost(h_ev,h_e,vcount,h_ee,h_mark,ecount,tid);
		}
		
		free(h_ev);
		free(h_e);
		free(h_ee);
		free(h_mark);
		
}


extern "C"
void	markContigStart(EulerEdge * d_ee, unsigned char * d_contigStart, unsigned int ecount){
	
	dim3 grid;
	dim3 block;
	hipMemset(d_contigStart,1,ecount);
	getOptimalLaunchConfiguration(ecount,&grid,&block);
	identifyContigStart<<<grid,block>>>(d_ee,d_contigStart,ecount);
	hipDeviceSynchronize();
	CheckCUDAError();

}
extern "C" 
void executeSwipeDevice(EulerVertex * d_ev,unsigned int * d_e, unsigned int vcount, EulerEdge * d_ee, unsigned int ecount, CircuitEdge * d_cg_edge,unsigned int cg_edgeCount , unsigned int * d_tree,unsigned int treeCount){
	dim3 grid ;
	dim3 block;

	unsigned int * d_mark;
	allocateMemory((void**) &d_mark, ecount* sizeof(unsigned int));

	hipMemset(d_mark,1,ecount* sizeof(unsigned int));
	getOptimalLaunchConfiguration(treeCount,&grid,&block);
	logMessage(LOG_LVL_DETAIL,"kernel: markSpanningEulerEdges");
	markSpanningEulerEdges<<<grid,block>>>(d_ee, d_mark , ecount,d_cg_edge,cg_edgeCount,d_tree, treeCount);
	hipDeviceSynchronize();
	CheckCUDAError();

	//DEBUG_CALL(executeSwipeHostLaunch(d_ev,d_e,vcount,d_ee,d_mark,ecount));
	getOptimalLaunchConfiguration(vcount,&grid,&block);
	logMessage(LOG_LVL_DETAIL,"kernel: executeSwipe");
	executeSwipe<<<grid,block>>>(d_ev,d_e,vcount , d_ee, d_mark,ecount);
	hipDeviceSynchronize();
	CheckCUDAError();
	
	//printData(d_ev,vcount,d_ee,ecount);
	deallocateMemory(d_mark);

}

/**ok ! this is not something pleasent to the eyes :-\*/
inline bool edgeComp(CircuitEdge a, CircuitEdge b){
	if(a.c1<b.c1) { return true;}
	else if(a.c1==b.c1){
		if(a.c2<b.c2){	return true;	} 
		else if (a.c2==b.c2){
			if(a.e1<b.e1) {return true;}
			else if(a.e1==b.e1){
				if(a.e2<b.e2) {return true;}
				else return false;
			} else return false;
		} else return false;		
	}else return false;
}
extern "C"
void findEulerDevice(EulerVertex * d_ev,unsigned int * d_l, unsigned int * d_e, unsigned int vcount,EulerEdge * d_ee,unsigned int ecount,CircuitEdge ** d_cg_edge, unsigned int * cg_edgeCount,unsigned int * cg_vertexCount, unsigned int kmerLength){


	Vertex * d_v=NULL;
	unsigned int * d_D;
	unsigned int * d_C;
	unsigned int * d_cg_offset;
	unsigned int * d_cedgeCount;
	unsigned int * d_cv;
	unsigned int * d_cg_edge_start;


	
	dim3 grid;
	dim3 block;
	
	allocateMemory((void**) &d_v, ecount* sizeof(Vertex));
		
	//step 1:
	// assign sucessors
	getOptimalLaunchConfiguration(vcount,&grid,&block);
	logMessage(LOG_LVL_DETAIL,"kernel: assignSuccessor");
	assignSuccessor<<<grid,block>>>(d_ev,d_l,d_e,vcount,d_ee,ecount);
	hipDeviceSynchronize();
	CheckCUDAError();
	
	//validateSuccessors(d_ee,ecount);
	
	//printDebruijnGraph(d_ev,vcount,d_l,d_e,d_ee,ecount,kmerLength,0);

	//step 2 successor graph
	//constructSuccessorGraph P1
	getOptimalLaunchConfiguration(ecount,&grid,&block);
	logMessage(LOG_LVL_DETAIL,"kernel: constructSuccessorGraph P1");
	constructSuccessorGraphP1<<<grid,block>>>(d_ee,d_v,ecount);
	hipDeviceSynchronize();
	CheckCUDAError();
	// printSuccessorGraph( d_v , ecount);
	
	/* synchronize */
	logMessage(LOG_LVL_DETAIL,"kernel: constructSuccessorGraph P2");
	constructSuccessorGraphP2<<<grid,block>>>(d_ee,d_v,ecount);
	hipDeviceSynchronize();
	CheckCUDAError();

	 //printSuccessorGraph( d_v , ecount);

	//step 3findComponent
	allocateMemory((void**) &d_D, ecount * sizeof(unsigned int));
	findComponentDevice(d_v,&d_D,ecount);
	

	//step 4 circuit graph construction
	//step 4.a  vertex calculation
	allocateMemory((void**) &d_C, ecount * sizeof(unsigned int));
	getOptimalLaunchConfiguration(ecount,&grid,&block);
	logMessage(LOG_LVL_DETAIL,"kernel: calculateCircuitGraphVertexData");
	calculateCircuitGraphVertexData<<<grid,block>>>( d_D,d_C,ecount);
	hipDeviceSynchronize();
	CheckCUDAError();
	//printData(d_C,ecount);

	//step 4.b offset calculation .find prefix sum 
	CUDPPConfiguration config;
	config.op = CUDPP_ADD;
	config.datatype = CUDPP_UINT;
    	config.algorithm = CUDPP_SCAN;
    	config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;
    
    	CUDPPHandle scanplan = 0;
    	CUDPPResult result = cudppPlan(&scanplan, config,ecount, 1, 0);	
	
	// Run the scan
	allocateMemory((void**) &d_cg_offset, ecount * sizeof(unsigned int));
    	cudppScan(scanplan, d_cg_offset, d_C, ecount);
	cudppDestroyPlan(scanplan);
	
	//printData(d_cg_offset,ecount);

	//step 4.c create circuitGraph
	unsigned int buffer[2];
	readData(buffer,d_cg_offset+ecount-1,1,sizeof(unsigned int));
	readData(buffer+1,d_C+ecount-1,1,sizeof(unsigned int));
	unsigned int circuitVertexSize=buffer[0]+buffer[1];
	*cg_vertexCount=circuitVertexSize;
	logMessage(LOG_LVL_MSG,"#Circuit Graph Vertex : %d",circuitVertexSize);
	allocateMemory( (void**) &d_cv, circuitVertexSize * sizeof(unsigned int));
	getOptimalLaunchConfiguration(ecount,&grid,&block);
	logMessage(LOG_LVL_DETAIL,"kernel: constructCircuitGraphVertex");
	constructCircuitGraphVertex<<<grid,block>>>(d_C,d_cg_offset,ecount, d_cv, circuitVertexSize);
	hipDeviceSynchronize();
	CheckCUDAError();
//	printData(d_cv,circuitVertexSize);

	if(circuitVertexSize>1){
		//step 4.d calculate edge information 
		allocateMemory((void**) &d_cedgeCount, circuitVertexSize * sizeof(unsigned int ));
		getOptimalLaunchConfiguration(vcount,&grid,&block);
		calculateCircuitGraphEdgeData<<<grid,block>>>(d_ev,d_e,vcount , d_D,d_cg_offset, ecount, d_cedgeCount/*, circuitVertexSize*/);
		hipDeviceSynchronize();
		CheckCUDAError();
		
		//printData(d_cedgeCount,circuitVertexSize);

		//step 4.e calculate edge offsets
		config.op = CUDPP_ADD;
		config.datatype = CUDPP_UINT;
		config.algorithm = CUDPP_SCAN;
		config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;    
		scanplan = 0;
		result = cudppPlan(&scanplan, config,ecount, 1, 0);		
		// Run the scan
		allocateMemory((void**) &d_cg_edge_start, circuitVertexSize * sizeof(unsigned int));
		cudppScan(scanplan, d_cg_edge_start, d_cedgeCount, circuitVertexSize);
		cudppDestroyPlan(scanplan);	
		//printData(d_cg_edge_start,circuitVertexSize);

		//step 4.f construct edges
		readData(buffer,d_cg_edge_start+circuitVertexSize-1,1,sizeof(unsigned int));
		readData(buffer+1,d_cedgeCount+circuitVertexSize-1,1,sizeof(unsigned int));
		unsigned int circuitGraphEdgeCount=buffer[0]+buffer[1];
		*cg_edgeCount=circuitGraphEdgeCount;
		logMessage(LOG_LVL_MSG,"#Circuit Graph Edges : %d\n",circuitGraphEdgeCount);

		
		allocateMemory((void**) d_cg_edge, circuitGraphEdgeCount * sizeof(CircuitEdge));
		//unsigned int * h_cedgeCount=NULL;
	//	h_cedgeCount = (unsigned int *) malloc(circuitVertexSize*sizeof(unsigned int));
	//	checkCudaErrors( hipMemcpy(h_cedgeCount, d_cedgeCount, circuitVertexSize*sizeof(unsigned int), hipMemcpyDeviceToHost));
		getOptimalLaunchConfiguration(vcount,&grid,&block);
		logMessage(LOG_LVL_DETAIL,"kernel: assignCircuitGraphEdgeData");
		assignCircuitGraphEdgeData<<<grid,block>>>(d_ev,d_e, vcount,d_D,d_cg_offset,ecount, d_cg_edge_start,d_cedgeCount, circuitVertexSize, *d_cg_edge, circuitGraphEdgeCount);
		hipDeviceSynchronize();
		CheckCUDAError();
		
	//	checkCudaErrors( hipMemcpy(d_cedgeCount,h_cedgeCount, circuitVertexSize*sizeof(unsigned int), hipMemcpyHostToDevice));
	//	free(h_cedgeCount);

		/**try1***/
/*		getOptimalLaunchConfigCustomized(circuitVertexSize,&grid,&block,1);
		for(unsigned char radix=0;radix<3;radix++){
			sortCircuitGraphEdgeData2<<<grid,block>>>(d_cg_edge_start,d_cedgeCount, circuitVertexSize, *d_cg_edge,radix);
			hipDeviceSynchronize();
			CheckCUDAError();
		}
*/		 
		/**try 2***/ 
/*		getOptimalLaunchConfigCustomized(circuitVertexSize,&grid,&block,1);
		unsigned short * d_mark;
		unsigned short * d_t2;
		allocateMemory((void**),d_mark,circuitGraphEdgeCount*sizeof(unsigned short));
		allocateMemory((void**),d_t1,circuitGraphEdgeCount*sizeof(unsigned short));
		getOptimalLaunchConfiguration(CircuitVertexSize,&grid,&block);
		markSegments<<<grid,block>>>(d_mark,circuitGraphEdgeCount,d_cg_edge_start,d_cedgeCount,circuitVertexSize);
		unsigned int thread=1;
		for(unsigned char radix=2;radix>=0;radix--){
			getOptimalLaunchConfigCustomized(circuitVertexSize,&grid,&block,threads);
			sortCircuitGraphEdgeData3<<<grid,block>>>(d_cg_edge_start,d_cedgeCount, circuitVertexSize, *d_cg_edge,d_mark,circuitGraphEdgeCount,radix);
			hipDeviceSynchronize();
			CheckCUDAError();
			thread++;
		}
		
		deallocateMemory(d_mark);
		deallocateMemory(d_t2);*/


		//*** try 3 cpu sorting**/
		CircuitEdge * h_cg_edge=(CircuitEdge *)malloc(circuitGraphEdgeCount*sizeof(CircuitEdge));
		checkCudaErrors( hipMemcpy(h_cg_edge,*d_cg_edge, circuitGraphEdgeCount*sizeof(CircuitEdge), hipMemcpyDeviceToHost));
		std::sort(h_cg_edge,h_cg_edge+circuitGraphEdgeCount,edgeComp);
		checkCudaErrors( hipMemcpy(*d_cg_edge,h_cg_edge, circuitGraphEdgeCount*sizeof(CircuitEdge), hipMemcpyHostToDevice));		
		free(h_cg_edge);	
	//	printCircuitGraph(*d_cg_edge,circuitGraphEdgeCount);	
		deallocateMemory(d_cg_edge_start);
		deallocateMemory(d_cedgeCount);
	}
	else{
		*cg_edgeCount=0;
	}
	//printData(*d_cg_edge,circuitGraphEdgeCount);
	
	/*****/
	//step 6 swipe execution

	
	deallocateMemory(d_cv);
	deallocateMemory(d_cg_offset);
	deallocateMemory(d_C);
	deallocateMemory(d_D);
	deallocateMemory(d_v);

	

	/*****/
	//calcuate contig start
	/***/

	
}
/*
extern "C"
void findEulerGold(EulerVertex * h_ev,
			unsigned int * h_l, 
			unsigned int * h_e, 
			unsigned int vcount,
			EulerEdge * h_ee,
			unsigned int ecount,
			unsigned int kmerLength){




	//find start vertices.
	unsigned char vertexMap;

	vertexMap=(unsigned char *) malloc(sizeof(unsigned char)*vcount);
	
	for(unsigned int i=0; i<vcount; i++){
		if(h_ev[i].ecount>h_ev[i].lcount){
		 	vertexMap[i]=1; //SINK
		}
		else if(h_ev[i].lcount>h_ev[i].ecount){
			vertexMap[i]=2; //SOURCE
		}
		else vertexMap[i]=0;
		// USED=100;
	
	}
	

	free(vertexMap);
		
}

*/
